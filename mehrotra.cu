#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hip/hip_runtime_api.h>

using namespace std;
#define CUDIE(result) {\
        hipError_t e = (result);\
        if (e != hipSuccess) {\
            std::cerr << __FILE__ << ":" << __LINE__;\
            std::cerr << " CUDA Runtime Error: " << hipGetErrorString(e) << "\n";\
            exit((int)e);\
        }}

#define CUDIE0() CUDIE(hipGetLastError())

#define THREADS_PER_BLOCK 256

void printVectorfloat(const float *V, int m);
void sparesify(float *M, int m, int n, float *A, int *IA, int *JA);
int findNNZ(const float *M, int N);
void printMatrix(const float *A, int nr_rows_A, int nr_cols_A);
void printVector(const int *V, int m);
void writeVec(float *V, int n, string fname);    
/*
* Batched Dot product of 2 Dense Vectors
* Each Block works on a single batch of vectors
* Number of Blocks must be >= Number of Batches (no_batch) 
*/
/*
* Batched Dot product of 2 Dense Vectors
* Each Block works on a single batch of vectors
* Number of Blocks must be >= Number of Batches (no_batch) 
*/
__global__ void dot(float *x_b, float *s_b, float *mu, int n, int no_batch)
{
    __shared__ float temp[THREADS_PER_BLOCK];
    temp[threadIdx.x] = 0.0;
    int batch;
    if (blockIdx.x < no_batch)
        batch = blockIdx.x;
    else
        return;
    float *x = &x_b[batch* n];
    float *s = &s_b[batch * n];   
    
    for (int i = threadIdx.x; i < n; i += blockDim.x)
        temp[i%THREADS_PER_BLOCK] += x[i] * s[i];
    __syncthreads();

    if (threadIdx.x == 0)
    {
        mu[batch] = 0;
        for (int i = 0; i < THREADS_PER_BLOCK; i++)
            mu[batch] += temp[i]/n;
    }
}

/*
* Multiplication of a CSR Sparse Matrix with a Vector
*/
__global__ void csr_mul_Av(float *csrValA, int *csrRowA,\
             int *csrColA, float *B_b, float *res_b, int m, int n, int nnz, int no_batch)
{  
    int batch;
    __shared__ float sum[THREADS_PER_BLOCK];
    __shared__ int i[THREADS_PER_BLOCK];
    if (blockIdx.x < no_batch)
        batch = blockIdx.x;
    else 
        return;

    float *B = &B_b[batch * n];
    float *res = &res_b[batch * m];

    for (int j = threadIdx.x; j < m; j += blockDim.x)
    {
        sum[threadIdx.x] = 0.0;
        for (i[threadIdx.x] = csrRowA[j] - 1; i[threadIdx.x] < csrRowA[j + 1] - 1; i[threadIdx.x]++)
        {
            sum[threadIdx.x] += csrValA[i[threadIdx.x]] * B[csrColA[i[threadIdx.x]] - 1]; 
        }     
        res[j] = sum[threadIdx.x];
    }  
}

/*
* Multiplication of CSR Matrix with a Diagonal Matrix (Batched)
* A is fixed, d (Diagonal Elements of Diagonal Matrix) varies
* if (div == 0) A * d
* else A * d / y
*/
__global__ void csr_diag_matmul(float *csrValA, int *csrRowA, int *csrColA, float *csrValres_b,\
                    float *d_b, float *y_b, int div, int m, int n, int nnz, int no_batch)
{
    int batch;
    if (blockIdx.x < no_batch)
        batch = blockIdx.x;
    else
        return;

    float *d = &d_b[batch * n];
    float *y = &y_b[batch * n];
    float *csrValRes = &csrValres_b[batch * nnz];
    float zy = 1.0;
    for (int j = threadIdx.x; j < nnz; j += blockDim.x)
    {
        if (div == 0)
            csrValRes[j] = csrValA[j] * d[csrColA[j] - 1];
        else if (div == 1)
        {
            if (abs(y[csrColA[j] - 1]) < 1e-9)
                zy = 1e-9;
            else
                zy = y[csrColA[j] - 1];
            csrValRes[j] = csrValA[j] / zy;
        }
        else
            csrValRes[j] = csrValA[j] * d[csrColA[j] - 1] / y[csrColA[j] - 1];
    }
}

/*
* Multiplication of Diagonal Matrix with vector 
* Diagonal Matrix is input as a Dense Vector of Diagonal Elements
* type = 0 for d1*x/d2, type = 1 for x/d2
*/
__global__ void diag_vector_mul(float *diag_b1, float *diag_b2, float *x_b, float *res_b, \
                                int n, int type, int no_batch)
{
    int batch;
    if (blockIdx.x < no_batch)
        batch = blockIdx.x;
    else 
        return;
    
    float *diag1 = &diag_b1[n * batch];
    float *diag2 = &diag_b2[n * batch];
    float *x = &x_b[n * batch];
    float *res = &res_b[n * batch];
    for(int i = threadIdx.x; i < n; i += blockDim.x)
    {
        float di;
        if (abs(diag2[i]) < 1e-9)
            di = 1e-9;
        else
            di = diag2[i];
        if (type == 0)
            res[i] = diag1[i] * x[i] / di;
        else    
            res[i] = x[i] / di;
    }
}
/*
* Batched Vector Addition of 3 vectors at a time
* Each Block works on a single batch of vectors
* Number of Blocks must be >= Number of Batches (no_batch)
*/
__global__ void vector_add(float *x_b, float *y_b, float *z_b, float *res_b, float a1, float a2, \
                        float a3, int n, int no_batch)
{
    int batch;
    if (blockIdx.x < no_batch)
        batch = blockIdx.x;
    else    
        return;
    float *x = &x_b[batch * n];
    float *y = &y_b[batch * n];
    float *z = &z_b[batch * n];
    float *res = &res_b[batch * n];

    for (int i = threadIdx.x; i < n; i += blockDim.x)
        res[i] = a1 * x[i] + a2 * y[i] + a3 * z[i];
}

/*
* Batched Vector Element Wise Multiplication of 
* 2 vectors-> res[i] = a1*mu + a2*x[i]*y[i] 
* Each Block works on a single batch of vectors
* Number of Blocks must be >= Number of Batches (no_batch)
*/
__global__ void vector_mul(float *x_b, float *y_b, float *mu, float *res_b,\
                            float a1, float a2, int n, int no_batch)
{
    int batch;
    if (blockIdx.x < no_batch)
        batch = blockIdx.x;
    else
        return;
    float *x = &x_b[batch * n];
    float *y = &y_b[batch * n];
    float *res = &res_b[batch * n];

    for (int i = threadIdx.x; i < n; i += blockDim.x)
        res[i] = a1 * mu[batch] + a2 * x[i] * y[i]; 
}

/*
*
*/
__global__ void find_update_param(float *x_b, float *s_b, float *ap_aff, \
                    float *ad_aff, float *dx_aff_b, float *ds_aff_b, int n, int no_batch)
{
    int batch;
    if(blockIdx.x < no_batch)
        batch = blockIdx.x;
    else
        return;

    if (threadIdx.x == 0)
    {
        float *x = &x_b[n * batch];
        float *dx_aff = &dx_aff_b[n * batch];
        float *s = &s_b[n * batch];
        float *ds_aff = &ds_aff_b[n * batch];
        for (int i = 0; i < n; i++)
        {
            if (dx_aff[i] < 0)
                ap_aff[batch] = min(ap_aff[batch], -0.9*x[i]/dx_aff[i]);
            if (ds_aff[i] < 0)
                ad_aff[batch] = min(ad_aff[batch], -0.9*s[i]/ds_aff[i]);
        }
    }
}

/*
*
*/
__global__ void update_vars(float *x_b, float *s_b, float *y_b, float *dx_aff_b,\
            float *ds_aff_b, float *dy_aff_b, float *ap_aff, float *ad_aff, \
            int m, int n, int no_batch)
{
    int batch;
    if (blockIdx.x < no_batch)
        batch = blockIdx.x;
    else    
        return;
    
    float *x = &x_b[n * batch];
    float *y = &y_b[m * batch];
    float *s = &s_b[n * batch];
    float *dx_aff = &dx_aff_b[n * batch];
    float *dy_aff = &dy_aff_b[m * batch];
    float *ds_aff = &ds_aff_b[n * batch];

    for (int i = threadIdx.x; i < n; i += blockDim.x)
    {
        x[i] = x[i] + ap_aff[batch] * dx_aff[i];
        s[i] = s[i] + ad_aff[batch] * ds_aff[i];
        if (i < m)
            y[i] = y[i] + ad_aff[batch] * dy_aff[i];
    }
}

/*
*
*/
__global__ void ADAt(float *csrValA, int *csrRowA, int *csrColA,\
            float *csrValADA_b, int *csrRowADA_b, int *csrColADA_b, float *x_b, \
            float *y_b, int m, int n, int nnz, int nnzAAt, int no_batch)
{
    int batch;
    if (blockIdx.x < no_batch)
        batch = blockIdx.x;
    else
        return;
    float *x = &x_b[batch * n];
    float *y = &y_b[batch * n];
    int *csrRowADA = &csrRowADA_b[batch * (m+1)];
    int *csrColADA = &csrColADA_b[batch * nnzAAt];
    float *csrValADA = &csrValADA_b[batch * nnzAAt];

    for (int i = threadIdx.x; i < m; i += blockDim.x)
    {
        for (int j = csrRowADA[i] - 1; j < csrRowADA[i+1] - 1; j++)
        {
            int k = csrColADA[j] - 1;
            float sum = 0.0;
            for (int a = csrRowA[i] - 1; a < csrRowA[i + 1] - 1; a++)
            {
                for (int b = csrRowA[k] - 1; b < csrRowA[k+1]-1; b++)
                {
                    if (csrColA[a] == csrColA[b])
                    {
                        float zy = y[csrColA[a]-1];
                        if (abs(zy) < 1e-9)
                            zy = 1e-9;
                        sum += csrValA[a] * csrValA[b] * x[csrColA[a]-1]/zy;
                    }
                }
            }
            csrValADA[j] = sum;
        }
    }
}


void AAtransposeCPU(float *csrValA, int *csrRowA, int *csrColA,\
            float *csrValADA, int *csrRowADA, int *csrColADA, int m, int n, int nnz)
{
    int nnzRes = 0;
    for (int j = 0; j < m; j++)
    {
        for (int k = 0; k < m; k++)
        {
            float sum = 0.0;
            for (int i = csrRowA[j] - 1; i < csrRowA[j+1] - 1; i++)
            {
                for (int m = csrRowA[k] - 1; m < csrRowA[k+1] - 1; m++)
                {
                    if (csrColA[i] == csrColA[m]) 
                    {
                        sum += csrValA[i] * csrValA[m];
                    }
                }
            }
            if (sum != 0.0)
            {
                csrValADA[nnzRes] = sum;
                csrColADA[nnzRes] = k + 1;
                nnzRes++;
            }
        }
        csrRowADA[j + 1] = nnzRes + 1;
    }
    
    csrRowADA[0] = 1;
    //for (int i = 1; i <= m; i++)
      //  csrRowADA[i] = csrRowADA[i-1] + csrRowADA[i];
}

/*void initilaize_solver(cusolverSpHandle_t *cusolverH, cusparseMatDescr_t *descrA, \
                        csrqrInfo_t *info)
{
    cusolverStatus_t cusolver_status = CUSOLVER_STATUS_SUCCESS;
    cusparseStatus_t cusparse_status = CUSPARSE_STATUS_SUCCESS;

    cusolver_status = cusolverSpCreate(cusolverH);
    assert (cusolver_status == CUSOLVER_STATUS_SUCCESS);
    cusparse_status = cusparseCreateMatDescr(descrA);
    assert(cusparse_status == CUSPARSE_STATUS_SUCCESS);
    cusparseSetMatType(*descrA, CUSPARSE_MATRIX_TYPE_GENERAL);
    cusparseSetMatIndexBase(*descrA, CUSPARSE_INDEX_BASE_ONE); // base-1
    cusolver_status = cusolverSpCreateCsrqrInfo(info);
    assert(cusolver_status == CUSOLVER_STATUS_SUCCESS);
}*/

int main()
{
    int m = 2, n = 4, batches = 3, iter = 10, nnz;
    float *A,  *csrValA,  *csrValADA;
    int *csrColA, *csrRowA, *csrColADA, *csrRowADA;
    int *csrColAT, *csrRowAT;// *csrColAS, *csrRowAS;
    float *csrValAT, *csrValAS, *csrValAD;
    float *x, *y, *s, *c, *b, *dx_aff, *dy_aff, *ds_aff, *rd, *rp, *rc, *v;   
    float *d_i1, *d_i2, *d_i3, *d_i4;
    //float sigma = 0.8;
    float *ap_aff, *ad_aff, *mu_aff, *mu, *cost;

    hipsolverSpHandle_t cusolverH = NULL;
    // GPU does batch QR
    csrqrInfo_t info = NULL;
    hipsparseMatDescr_t descrA = NULL;
    hipsparseStatus_t cusparse_status = HIPSPARSE_STATUS_SUCCESS;
    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    size_t size_qr = 0;
    size_t size_internal = 0;
    void *buffer_qr = NULL; // working space for numerical factorization
    hipError_t cudaStat1 = hipSuccess;
    
    // Initialize A
    A = (float*)malloc(m* n * sizeof(float));
    for (int i = 0; i < m*n; i++)
        A[i] = 0;
    A[0] = 1, A[3] = 1, A[4] = 1, A[7] = 1;
    nnz = findNNZ(A, m*n);
    printf("Non Zero Elements: %d\n", nnz);

    hipMallocManaged(&csrValA, sizeof(float)*nnz);
    hipMallocManaged(&csrRowA, sizeof(int)*(m+1));
    hipMallocManaged(&csrColA, sizeof(int)*nnz);
    hipMallocManaged(&csrValAT, sizeof(float)*nnz);
    hipMallocManaged(&csrRowAT, sizeof(int)*(m+1));
    hipMallocManaged(&csrColAT, sizeof(int)*nnz);
    hipMallocManaged(&csrValAS, sizeof(float)*nnz*batches);
    hipMallocManaged(&csrValAD, sizeof(float)*nnz*batches);
    hipMallocManaged(&csrValADA, sizeof(float)*2*batches);
    hipMallocManaged(&csrRowADA, sizeof(int)*(m+1)*batches);
    hipMallocManaged(&csrColADA, sizeof(int)*2*batches);
    hipMallocManaged(&x, sizeof(float)* n * batches);
    hipMallocManaged(&s, sizeof(float)* n * batches);
    hipMallocManaged(&y, sizeof(float)* m * batches);
    hipMallocManaged(&c, sizeof(float)* n * batches);
    hipMallocManaged(&b, sizeof(float)* m * batches);
    hipMallocManaged(&dx_aff, sizeof(float) * n *batches);
    hipMallocManaged(&dy_aff, sizeof(float) * m * batches);
    hipMallocManaged(&d_i1, sizeof(float) * m * batches);
    hipMallocManaged(&d_i2, sizeof(float) * m * batches);
    hipMallocManaged(&d_i3, sizeof(float) * m * batches);
    hipMallocManaged(&ds_aff, sizeof(float) * n * batches);
    hipMallocManaged(&d_i4, sizeof(float) * n * batches);
    hipMallocManaged(&rd, sizeof(float)* n * batches);
    hipMallocManaged(&rp, sizeof(float)* m * batches);
    hipMallocManaged(&rc, sizeof(float)* n * batches);
    hipMallocManaged(&v, sizeof(float)* n * batches);
    hipMallocManaged(&ap_aff, sizeof(float) * batches);
    hipMallocManaged(&ad_aff, sizeof(float) * batches);
    hipMallocManaged(&mu_aff, sizeof(float) * batches);
    hipMallocManaged(&mu, sizeof(float) * batches);
    hipMallocManaged(&cost, sizeof(float) * batches);

    /*Initialization of the Problem*/
    float ib[2] = {1, 2}, ic[4] = {-1, -1, 1, 1};
    float ix[4] = {0.8, 0.1, 0.2, 1.9}, is[4] = {0.1, 0.2, 2.1, 2.2}, iy[2] = {-1.1, -1.2};
    
    for (int i = 0; i < batches; i++) {
        for (int j = 0; j < m; j++) {
            b[m*i + j] = ib[j];
            y[m*i + j] = iy[j];
        }
    }        
    for (int i = 0; i < batches; i++) {
        for (int j = 0; j < n; j++) {
            c[n*i + j] = ic[j];
            x[n*i + j] = ix[j];
            s[n*i + j] = is[j];
        }
    }
    sparesify(A, m, n, csrValA, csrRowA, csrColA);
    for (int i = 0; i < m*n; i++)
        A[i] = 0;
    A[0] = 1, A[2] = 1, A[5] = 1, A[7] = 1;
    sparesify(A, n, m, csrValAT, csrRowAT, csrColAT);
    int nnzAAt = 2;
    for (int i = 0; i < batches; i++)
    {
        AAtransposeCPU(csrValA, csrRowA, csrColA, &csrValADA[nnzAAt*i],\
                    &csrRowADA[(m+1)*i], &csrColADA[nnzAAt*i], m, n, nnz);
    }   
    printVectorfloat(csrValADA, nnzAAt * batches);
    printVector(csrRowADA, (m + 1)*batches);
    printVector(csrColADA, nnzAAt * batches);

    /*---Initialization*/

    //printVectorfloat(x, n*batches);
    /*Main Loop for IPM iterations*/
    for (int i = 0; i < iter; i++)
    {
        dot<<<batches, THREADS_PER_BLOCK>>>(x, s, mu, n, batches);
        hipDeviceSynchronize();
        csr_mul_Av<<<batches, THREADS_PER_BLOCK>>>(csrValAT, csrRowAT, csrColAT, y, rd,\
                                                    n, m, nnz, batches);
        hipDeviceSynchronize();
        CUDIE0();
        vector_add<<<batches, THREADS_PER_BLOCK>>>(c, s, rd, rd, 1, -1, -1, n, batches);
        hipDeviceSynchronize();
        csr_mul_Av<<<batches, THREADS_PER_BLOCK>>>(csrValA, csrRowA, csrColA, x, rp,\
                                                    m, n, nnz, batches);
        hipDeviceSynchronize();
        vector_add<<<batches, THREADS_PER_BLOCK>>>(b, rp, rp, rp, 1, -1, 0, m, batches);
        hipDeviceSynchronize();
        vector_mul<<<batches, THREADS_PER_BLOCK>>>(x, s, mu, rc, 0, -1, n, batches);
        hipDeviceSynchronize();
        ADAt<<<batches, THREADS_PER_BLOCK>>>(csrValA, csrRowA, csrColA, csrValADA,\
                                csrRowADA, csrColADA, x, s, m, n, nnz, 2, batches);
        hipDeviceSynchronize();
        csr_diag_matmul<<<batches, THREADS_PER_BLOCK>>>(csrValA, csrRowA, csrColA,\
                                 csrValAS, s, s, 1, m, n, nnz, batches);
        hipDeviceSynchronize();
        csr_mul_Av<<<batches, THREADS_PER_BLOCK>>>(csrValAS, csrRowA, csrColA, rc, d_i1,\
                                                    m, n, nnz, batches);
        hipDeviceSynchronize();
        csr_diag_matmul<<<batches, THREADS_PER_BLOCK>>>(csrValA, csrRowA, csrColA,\
                                 csrValAD, x, s, 2, m, n, nnz, batches);
        hipDeviceSynchronize();
        csr_mul_Av<<<batches, THREADS_PER_BLOCK>>>(csrValAD, csrRowA, csrColA, rd, d_i2,\
                                                    m, n, nnz, batches);
        hipDeviceSynchronize();
        vector_add<<<batches, THREADS_PER_BLOCK>>>(d_i1, d_i2, rp, d_i3, -1, 1, 1, m, batches);
        hipDeviceSynchronize();

        //Solve for dy_aff
        // step 2: create cusolver handle, qr info and matrix descriptor
        cusolver_status = hipsolverSpCreate(&cusolverH);
        assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);
        cusparse_status = hipsparseCreateMatDescr(&descrA);
        assert(cusparse_status == HIPSPARSE_STATUS_SUCCESS);
        hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE); // base-1
        cusolver_status = cusolverSpCreateCsrqrInfo(&info);
        assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);
        // step 4: symbolic analysis
        cusolver_status = cusolverSpXcsrqrAnalysisBatched(cusolverH, m, m, nnzAAt, \
        descrA, csrRowADA, csrColADA,info);
        assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);
        // step 5: prepare working space
        cusolver_status = cusolverSpScsrqrBufferInfoBatched(cusolverH, m, m, nnzAAt,\
        descrA, csrValADA, csrRowADA, csrColADA,\
        batches,\
        info,\
        &size_internal,\
        &size_qr);
        assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);
        printf("numerical factorization needs internal data %lld bytes\n",(long long)size_internal);
        printf("numerical factorization needs working space %lld bytes\n",(long long)size_qr);
        cudaStat1 = hipMalloc((void**)&buffer_qr, size_qr);
        assert(cudaStat1 == hipSuccess);
        // step 6: numerical factorization
        // assume device memory is big enough to compute all matrices.
        cusolver_status = cusolverSpScsrqrsvBatched(cusolverH, m, m, nnzAAt,\
        descrA, csrValADA, csrRowADA, csrColADA,\
        d_i3, dy_aff,\
        batches,\
        info,\
        buffer_qr);
        assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);
        const int baseA = (HIPSPARSE_INDEX_BASE_ONE == hipsparseGetMatIndexBase(descrA))? 1:0 ;
        for(int batchId = 0 ; batchId < batches; batchId++){
            // measure |bj - Aj*xj|
            float *csrValAj = csrValADA + batchId * nnzAAt;
            float *xj = dy_aff + batchId * m;
            float *bj = d_i3 + batchId * m;
            // sup| bj - Aj*xj|
            float sup_res = 0;
            for(int row = 0 ; row < m ; row++){
                const int start = csrRowADA[row ] - baseA;
                const int end = csrRowADA[row+1] - baseA;
                float Ax = 0.0; // Aj(row,:)*xj
                for(int colidx = start ; colidx < end ; colidx++){
                    const int col = csrColADA[colidx] - baseA;
                    const float Areg = csrValAj[colidx];
                    const float xreg = xj[col];
                    Ax = Ax + Areg * xreg;
                }
                float r = bj[row] - Ax;
                sup_res = (sup_res > fabs(r))? sup_res : fabs(r);
            }
                printf("batchId %d: sup|bj - Aj*xj| = %E \n", batchId, sup_res);
        }
        for(int batchId = 0 ; batchId < batches; batchId++){
            float *xj = dy_aff + batchId * m;
            for(int row = 0 ; row < m ; row++){
                printf("x%d[%d] = %E\n", batchId, row, xj[row]);
            }
            printf("\n");
        }

        csr_mul_Av<<<batches, THREADS_PER_BLOCK>>>(csrValAT, csrRowAT, csrColAT, dy_aff, ds_aff,\
                                                    n, m, nnz, batches);
        hipDeviceSynchronize();
        vector_add<<<batches, THREADS_PER_BLOCK>>>(ds_aff, rd, ds_aff, ds_aff, -1, 1, 0, n, batches);
        hipDeviceSynchronize();
        diag_vector_mul<<<batches, THREADS_PER_BLOCK>>>(s, s, rc, d_i4, n, 1, batches);
        hipDeviceSynchronize();
        diag_vector_mul<<<batches, THREADS_PER_BLOCK>>>(x, s, ds_aff, dx_aff, n, 0, batches);
        hipDeviceSynchronize();
        vector_add<<<batches, THREADS_PER_BLOCK>>>(dx_aff, d_i4, dx_aff, dx_aff, -1, 1, 0, n, batches);
        hipDeviceSynchronize();

        for (int j = 0; j < batches; j++)
        {
            ap_aff[j] = 1.0;
            ad_aff[j] = 1.0;
        }
        find_update_param<<<batches, THREADS_PER_BLOCK>>>(x, s, ap_aff, ad_aff, dx_aff,\
                                                ds_aff, n, batches);
        hipDeviceSynchronize();
        update_vars<<<batches, THREADS_PER_BLOCK>>>(x, s, y, dx_aff, ds_aff, dy_aff, ap_aff,\
                                ad_aff, m, n, batches);
        hipDeviceSynchronize();
        writeVec(x, n, "x_upd.txt");
        writeVec(s, n, "s_upd.txt");
        cout << "Cost: " << endl;
        dot<<<batches, THREADS_PER_BLOCK>>>(c, x, cost, n, batches);
        hipDeviceSynchronize();
        for (int j = 0; j < batches; j++)
            cout << cost[j]*n << " ";
        cout << endl;

    }
    //for (int i  = 0; i < batches; i++)
      //  cout << mu[i] << endl;
    /*for (int i = 0; i < batches; i++){
        for (int j = 0; j < n; j++)
            cout << rd[i*n + j] << " ";
        cout << endl;
    }
    printVectorfloat(csrValADA, 2*batches);
    printVectorfloat(d_i1, m*batches);
    printVectorfloat(d_i2, m*batches);
    printVectorfloat(d_i3, m*batches);*/
    printVectorfloat(x, n * batches);
}


void printVectorfloat(const float *V, int m)
{
    for (int i = 0; i < m; i++)
        std::cout << V[i] << " ";
    std::cout << std::endl;
}
// Generate the three vectors A, IA, JA 
void sparesify(float *M, int m, int n, float *A, int *IA, int *JA)
{
    //int m = M.size(), n = M[0].size();
    int i, j;
    //vi A;
    IA[0] = 1; // IA matrix has N+1 rows
    //vi JA;
    int NNZ = 0;
  
    for (i = 0; i < m; i++) {
        for (j = 0; j < n; j++) {
            if (M[i + m*j] != 0) {
                A[NNZ] = M[i + m*j];
                JA[NNZ] = j + 1;
  
                // Count Number of Non Zero 
                // Elements in row i
                NNZ++;
            }
        }
        IA[i + 1] = NNZ + 1;
    }
  
    printMatrix(M, m, n);
    printVectorfloat(A, NNZ);
    printVector(IA, m + 1);
    printVector(JA, NNZ);
}
//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
 void printMatrix(const float *A, int nr_rows_A, int nr_cols_A) {
 
     for(int i = 0; i < nr_rows_A; ++i){
         for(int j = 0; j < nr_cols_A; ++j){
             std::cout << A[j * nr_rows_A + i] << " ";
         }
         std::cout << std::endl;
     }
     std::cout << std::endl;
 }
  
// Utility Function to print A, IA, JA vectors
// with some decoration.
void printVector(const int *V, int m)
{
    for (int i = 0; i < m; i++)
        std::cout << V[i] << " ";
    std::cout << std::endl;
}

/*
* Find Number of Non Zero Elements in a Sparse Matrix
*/
int findNNZ(const float *M, int N)
{
    int nnz = 0;
    for (int i = 0;  i < N; i++)
    {
        if (M[i] != 0.0)
            nnz++;
    }

    return nnz;
}

void writeVec(float *V, int n, string fname)
{
    ofstream outdata;
    outdata.open(fname);
    for (int i = 0; i < n; i++)
    {
        outdata << V[i] << endl;
    }
    outdata.close();
}