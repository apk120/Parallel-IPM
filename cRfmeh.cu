#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hip/hip_runtime_api.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include "inc/helper_string.h"
#include "inc/helper_cusolver.h"
#include "inc/hip/hip_runtime_api.h"
#include <hipsparse.h>         // cusparseSpGEMM
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

using namespace std;
#define CUDIE(result) {\
        hipError_t e = (result);\
        if (e != hipSuccess) {\
            std::cerr << __FILE__ << ":" << __LINE__;\
            std::cerr << " CUDA Runtime Error: " << hipGetErrorString(e) << "\n";\
            exit((int)e);\
        }}

#define CUDIE0() CUDIE(hipGetLastError())


#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}
#define THREADS_PER_BLOCK 256

void printVectordouble(const double *V, int m);
// void sparesify(double *M, int m, int n, double *A, int *IA, int *JA);
int findNNZ(const double *M, int N);
void printMatrix(const double *A, int nr_rows_A, int nr_cols_A);
void printVector(const int *V, int m);
void getMatSize(string fname, int *rows, int *nnz);
void loadMat(double *A, int *IA, int *JA, string fname, int *rows, int *nnz);
void loadVec(double *V, string fname);
void writeVec(double *V, int n, string fname);
void writeVecInt(int *V, int n, string fname);
void createA(double *A, double *csrValA, int *csrRowA, int *csrColA, int m, int n, int nnz, int baseA);
int solve (int rowsA, int colsA, int N, int nnzA, int batchSize, int baseA, \
            double *h_csrValA, int *h_csrRowPtrA, int *h_csrColIndA, \
             double *x, double *b, struct testOpts opts);
int xsc(int *hA_csrOffsets, int *hA_columns, double *hA_values,int *hB_csrOffsets, int *hB_columns, double *hB_values, int *hC_csrOffsets, int *hC_columns, double *hC_values, int C_nnz, double *resC) ;

void parseCommandLineArguments(int argc, char *argv[], struct testOpts &opts);
/*
* Batched Dot product of 2 Dense Vectors
* Each Block works on a single batch of vectors
* Number of Blocks must be >= Number of Batches (no_batch) 
*/
__global__ void dot(double *x_b, double *s_b, double *mu, int n, int no_batch)
{
    __shared__ double temp[THREADS_PER_BLOCK];
    temp[threadIdx.x] = 0.0;
    int batch;
    if (blockIdx.x < no_batch)
        batch = blockIdx.x;
    else
        return;
    double *x = &x_b[batch* n];
    double *s = &s_b[batch * n];   
    
    for (int i = threadIdx.x; i < n; i += blockDim.x)
        temp[i%THREADS_PER_BLOCK] += x[i] * s[i];
    __syncthreads();

    if (threadIdx.x == 0)
    {
        mu[batch] = 0;
        for (int i = 0; i < THREADS_PER_BLOCK; i++)
            mu[batch] += temp[i]/n;
    }
}

/*
*
*/
__global__ void findmuAff(double *x_b, double *s_b, double *mu, double *ap_b, double *dx_b, double *ad_b, double *ds_b, int n, int no_batch)
{
    __shared__ double temp[THREADS_PER_BLOCK];
    temp[threadIdx.x] = 0.0;
    int batch;
    if (blockIdx.x < no_batch)
        batch = blockIdx.x;
    else
        return;
    double *x = &x_b[batch* n];
    double *s = &s_b[batch * n];   
    double *dx = &dx_b[batch * n];
    double *ds = &ds_b[batch * n];
    double ap = ap_b[batch];
    double ad = ad_b[batch];
    for (int i = threadIdx.x; i < n; i += blockDim.x)
        temp[i%THREADS_PER_BLOCK] += (x[i] + ap * dx[i])* (s[i] + ad * ds[i]);
    __syncthreads();

    if (threadIdx.x == 0)
    {
        mu[batch] = 0;
        for (int i = 0; i < THREADS_PER_BLOCK; i++)
            mu[batch] += temp[i]/n;
    }
}
/*
* Multiplication of a CSR Sparse Matrix with a Vector
*/
__global__ void csr_mul_Av(double *csrValA, int *csrRowA,\
             int *csrColA, double *B_b, double *res_b, int m, int n, int nnz, int no_batch, int baseA)
{  
    int batch;
    //__shared__ double sum[THREADS_PER_BLOCK];
    __shared__ int i[THREADS_PER_BLOCK];
    if (blockIdx.x < no_batch)
        batch = blockIdx.x;
    else 
        return;

    double *B = &B_b[batch * n];
    double *res = &res_b[batch * m];

    for (int j = threadIdx.x; j < m; j += blockDim.x)
    {
        double sum = 0.0;
        for (i[threadIdx.x] = csrRowA[j] - baseA; i[threadIdx.x] < csrRowA[j + 1] - baseA; i[threadIdx.x]++)
        {
            if (i[threadIdx.x] >= nnz)
                printf("wrrong, %d %d %d m = %d, n = %d\n", i[threadIdx.x], j, threadIdx.x, m, n);
            sum += csrValA[i[threadIdx.x]] * B[csrColA[i[threadIdx.x]] - baseA]; //B[]
        }     
        res[j] = sum;
    }  
}

/*
* Multiplication of CSR Matrix with a Diagonal Matrix (Batched)
* A is fixed, d (Diagonal Elements of Diagonal Matrix) varies
* if (div == 0) A * d
* else A * d / y
*/
__global__ void csr_diag_matmul(double *csrValA, int *csrRowA, int *csrColA, double *csrValres_b,\
                    double *d_b, double *y_b, int div, int m, int n, int nnz, int no_batch, int baseA)
{
    int batch;
    if (blockIdx.x < no_batch)
        batch = blockIdx.x;
    else
        return;

    double *d = &d_b[batch * n];
    double *y = &y_b[batch * n];
    double *csrValRes = &csrValres_b[batch * nnz];
    double zy = 1.0;
    for (int j = threadIdx.x; j < nnz; j += blockDim.x)
    {
        if (div == 0)
            csrValRes[j] = csrValA[j] * d[csrColA[j] - baseA];
        else if (div == 1)
        {
            double sign = 1.0;
            if (y[csrColA[j] - baseA] < 0)
                sign = -1.0;
            
            if (abs(y[csrColA[j] - baseA]) < 1e-9)
                zy = 1e-9*sign;
            else
                zy = y[csrColA[j] - baseA];
            csrValRes[j] = csrValA[j] / zy;
        }
        else
        {
            double sign = 1.0;
            if (y[csrColA[j] - baseA] < 0)
                sign = -1.0;
            
            if (abs(y[csrColA[j] - baseA]) < 1e-9)
                zy = 1e-9*sign;
            else
                zy = y[csrColA[j] - baseA];
            csrValRes[j] = csrValA[j] * (d[csrColA[j] - baseA]) / zy;//+1e-13
        }   
    }
}

/*
* Multiplication of Diagonal Matrix with vector 
* Diagonal Matrix is input as a Dense Vector of Diagonal Elements
* type = 0 for d1*x/d2, type = 1 for x/d2
*/
__global__ void diag_vector_mul(double *diag_b1, double *diag_b2, double *x_b, double *res_b, \
                                int n, int type, int no_batch)
{
    int batch;
    if (blockIdx.x < no_batch)
        batch = blockIdx.x;
    else 
        return;
    
    double *diag1 = &diag_b1[n * batch];
    double *diag2 = &diag_b2[n * batch];
    double *x = &x_b[n * batch];
    double *res = &res_b[n * batch];
    for(int i = threadIdx.x; i < n; i += blockDim.x)
    {
        double di;
        double sign = 1.0;
        if (diag2[i] < 0)
            sign = -1.0;
            
        if (abs(diag2[i]) < 1e-9)
            di = 1e-9*sign;
        else
            di = diag2[i];

        if (type == 0)
            res[i] = diag1[i] * x[i] / di;
        else    
            res[i] = x[i] / di;
    }
}
/*
* Batched Vector Addition of 3 vectors at a time
* Each Block works on a single batch of vectors
* Number of Blocks must be >= Number of Batches (no_batch)
*/
__global__ void vector_add(double *x_b, double *y_b, double *z_b, double *res_b, double a1, double a2, \
                        double a3, int n, int no_batch)
{
    int batch;
    if (blockIdx.x < no_batch)
        batch = blockIdx.x;
    else    
        return;
    double *x = &x_b[batch * n];
    double *y = &y_b[batch * n];
    double *z = &z_b[batch * n];
    double *res = &res_b[batch * n];

    for (int i = threadIdx.x; i < n; i += blockDim.x)
        res[i] = a1 * x[i] + a2 * y[i] + a3 * z[i];
}

/*
* Batched Vector Element Wise Multiplication of 
* 2 vectors-> res[i] = a1*mu + a2*x[i]*y[i] 
* Each Block works on a single batch of vectors
* Number of Blocks must be >= Number of Batches (no_batch)
*/
__global__ void vector_mul(double *x_b, double *y_b, double *mu, double *res_b,\
                            double a1, double a2, int n, int no_batch)
{
    int batch;
    if (blockIdx.x < no_batch)
        batch = blockIdx.x;
    else
        return;
    double *x = &x_b[batch * n];
    double *y = &y_b[batch * n];
    double *res = &res_b[batch * n];

    for (int i = threadIdx.x; i < n; i += blockDim.x)
        res[i] = a1 * mu[batch] + a2 * x[i] * y[i]; 
}

/*
*
*/
__global__ void find_update_param(double *x_b, double *s_b, double *ap_aff, \
                    double *ad_aff, double *dx_aff_b, double *ds_aff_b, int n, int no_batch)
{
    int batch;
    if(blockIdx.x < no_batch)
        batch = blockIdx.x;
    else
        return;

    if (threadIdx.x == 0)
    {
        double *x = &x_b[n * batch];
        double *dx_aff = &dx_aff_b[n * batch];
        double *s = &s_b[n * batch];
        double *ds_aff = &ds_aff_b[n * batch];
        for (int i = 0; i < n; i++)
        {
            if (dx_aff[i] < 0)
                ap_aff[batch] = min(ap_aff[batch], -0.9*x[i]/dx_aff[i]);
            if (ds_aff[i] < 0)
                ad_aff[batch] = min(ad_aff[batch], -0.9*s[i]/ds_aff[i]);
        }
    }
}

/*
*
*/
__global__ void update_vars(double *x_b, double *s_b, double *y_b, double *dx_aff_b,\
            double *ds_aff_b, double *dy_aff_b, double *ap_aff, double *ad_aff, \
            int m, int n, int no_batch)
{
    int batch;
    if (blockIdx.x < no_batch)
        batch = blockIdx.x;
    else    
        return;
    
    double *x = &x_b[n * batch];
    double *y = &y_b[m * batch];
    double *s = &s_b[n * batch];
    double *dx_aff = &dx_aff_b[n * batch];
    double *dy_aff = &dy_aff_b[m * batch];
    double *ds_aff = &ds_aff_b[n * batch];

    for (int i = threadIdx.x; i < n; i += blockDim.x)
    {
        x[i] = x[i] + ap_aff[batch] * dx_aff[i];
        s[i] = s[i] + ad_aff[batch] * ds_aff[i];
        if (i < m)
            y[i] = y[i] + ad_aff[batch] * dy_aff[i];
    }
}

/*
*
*/
__global__ void ADAt(double *csrValA, int *csrRowA, int *csrColA,\
            double *csrValADA_b, int *csrRowADA_b, int *csrColADA_b, double *x_b, \
            double *y_b, int m, int n, int nnz, int nnzAAt, int no_batch)
{
    int batch;
    __shared__ double sum[THREADS_PER_BLOCK];
    __shared__ int k[THREADS_PER_BLOCK];
    __shared__ int j[THREADS_PER_BLOCK];
    __shared__ int a[THREADS_PER_BLOCK];
    __shared__ int b[THREADS_PER_BLOCK];
    if (blockIdx.x < no_batch)
        batch = blockIdx.x;
    else
        return;
    double *x = &x_b[batch * n];
    double *y = &y_b[batch * n];
    int *csrRowADA = &csrRowADA_b[batch * (m+1)];
    int *csrColADA = &csrColADA_b[batch * nnzAAt];
    double *csrValADA = &csrValADA_b[batch * nnzAAt];

    for (int i = threadIdx.x; i < m; i += blockDim.x)
    {
        for (j[threadIdx.x] = csrRowADA[i] - 1; j[threadIdx.x] < csrRowADA[i+1] - 1; j[threadIdx.x]++)
        {
            k[threadIdx.x] = csrColADA[j[threadIdx.x]] - 1;
            sum[threadIdx.x] = 0.0;
            for (a[threadIdx.x] = csrRowA[i] - 1; a[threadIdx.x] < csrRowA[i + 1] - 1; a[threadIdx.x]++)
            {
                for (b[threadIdx.x] = csrRowA[k[threadIdx.x]] - 1; b[threadIdx.x] < csrRowA[k[threadIdx.x]+1]-1; b[threadIdx.x]++)
                {
                    if (csrColA[a[threadIdx.x]] == csrColA[b[threadIdx.x]])
                    {
                        double zy = y[csrColA[a[threadIdx.x]]-1];
                        double sign = 1.0;
                        if (zy < 0)
                            sign = -1.0;
                        if (abs(zy) < 1e-9)
                            zy = 1e-9 *sign;
                        sum[threadIdx.x] += csrValA[a[threadIdx.x]] * csrValA[b[threadIdx.x]] * x[csrColA[a[threadIdx.x]]-1]/zy;
                    }
                }
            }
            csrValADA[j[threadIdx.x]] = sum[threadIdx.x];
        }
    }
}

__global__ void ADAT(double *A, double *csrValAD_b, int *csrRowAD, int *csrColAD,\
            double *csrValADA_b, int *csrRowADA, int *csrColADA, double *x_b, \
            double *y_b, int m, int n, int nnz, int nnzAAt, int no_batch, int baseA)
{
    int batch;
    if (blockIdx.x < no_batch)
        batch = blockIdx.x;
    else
        return;
    //double *x = &x_b[batch * n];
    //double *y = &y_b[batch * n];
    double *csrValADA = &csrValADA_b[batch * nnzAAt];
    
    double *csrValAD = &csrValAD_b[batch * nnz];

    for (int i = threadIdx.x; i < m; i += blockDim.x)
    {
        for (int j = csrRowADA[i] - baseA; j < csrRowADA[i+1] - baseA; j++)
        {
            if (j < 0)
                printf("Error %d\n", j);
            int k = csrColADA[j] - baseA;
            double sum = 0.0;
            for (int a = csrRowAD[i]-baseA; a < csrRowAD[i+1]-baseA; a++)
            {
                // if (k + (csrColAD[a]-baseA)*m > no_batch*m*n)
                //     printf("Wrong %d batch %d a %d csrrCol %d m %d k %d i %d\n", k + (csrColAD[a]-baseA)*m, batch, a, csrColAD[a], m, k, i);
                if (csrColAD[a] < 0)
                    printf("Worng nnz %d\n", a);
                sum += csrValAD[a] * A[k + (csrColAD[a]-baseA)*m];
            }
            if (j >= nnzAAt)
                printf("Worng nnzAAt %d", j);
            csrValADA[j] = sum + 1e-8;
        }
    }
}

void AAtransposeCPU(double *csrValA, int *csrRowA, int *csrColA,\
            double *csrValADA, int *csrRowADA, int *csrColADA, int m, int n, int nnz)
{
    int nnzRes = 0;
    for (int j = 0; j < m; j++)
    {
        for (int k = 0; k < m; k++)
        {
            double sum = 0.0;
            for (int i = csrRowA[j] - 1; i < csrRowA[j+1] - 1; i++)
            {
                for (int m = csrRowA[k] - 1; m < csrRowA[k+1] - 1; m++)
                {
                    if (csrColA[i] == csrColA[m]) 
                    {
                        sum += csrValA[i] * csrValA[m];
                    }
                }
            }
            if (sum != 0.0)
            {
                csrValADA[nnzRes] = sum;
                csrColADA[nnzRes] = k + 1;
                nnzRes++;
            }
        }
        csrRowADA[j + 1] = nnzRes + 1;
    }
    
    csrRowADA[0] = 1;
    //for (int i = 1; i <= m; i++)
      //  csrRowADA[i] = csrRowADA[i-1] + csrRowADA[i];
}


int main(int argc, char *argv[])
{
    int m = 6590, n = 12879, batches = 8, iter = 50, nnz; // 6590, 12879, 2036312
    double *csrValA,  *csrValADA, *A;//*A
    int *csrColA, *csrRowA, *csrColADA, *csrRowADA, *csrColAAT, *csrRowAAT;
    int *csrColAT, *csrRowAT;// *csrColAS, *csrRowAS;
    double *csrValAT, *csrValAS, *csrValAD, *csrValAAT;
    double *x, *y, *s, *c, *b, *dx_aff, *dy_aff, *ds_aff, *rd, *rp, *rc, *v, *ds_cor, *dy_cor;   
    double *d_i1, *d_i2, *d_i3, *d_i4;
    //double sigma = 0.8;
    double *ap_aff, *ad_aff, *mu_aff, *mu, *cost, *sigma;
    int nnz_aat = 2042300;
    int base = 0;
    struct testOpts opts;
    double start_t, stop_t, time_preop, time_ada, time_post;
    double min_norm;
    int rankAt;
    int *perm;

    parseCommandLineArguments(argc, argv, opts);
    findCudaDevice(argc, (const char **)argv);

    string fname = "A_sparse.csv";
    getMatSize(fname, &m, &nnz);
    printf("Non Zero Elements: %d, rows: %d\n", nnz, m);

    hipMallocManaged(&A, sizeof(double)*m*n);
    CUDIE0();
    hipMallocManaged(&csrValA, sizeof(double)*nnz);
    hipMallocManaged(&csrRowA, sizeof(int)*(m+1));
    hipMallocManaged(&csrColA, sizeof(int)*nnz);
    hipMallocManaged(&csrValAT, sizeof(double)*nnz);
    hipMallocManaged(&csrRowAT, sizeof(int)*(n+1));
    hipMallocManaged(&csrColAT, sizeof(int)*nnz);
    hipMallocManaged(&csrValAS, sizeof(double)*nnz*batches);
    hipMallocManaged(&csrValAD, sizeof(double)*nnz*batches);
    hipMallocManaged(&csrValADA, sizeof(double)*nnz_aat*batches);
    hipMallocManaged(&csrRowADA, sizeof(int)*(m+1));
    hipMallocManaged(&csrColADA, sizeof(int)*nnz_aat);
    hipMallocManaged(&csrValAAT, sizeof(double)*nnz_aat);
    hipMallocManaged(&csrRowAAT, sizeof(int)*(m+1));
    hipMallocManaged(&csrColAAT, sizeof(int)*nnz_aat);
    hipMallocManaged(&x, sizeof(double)* n * batches);
    hipMallocManaged(&s, sizeof(double)* n * batches);
    hipMallocManaged(&y, sizeof(double)* m * batches);
    hipMallocManaged(&c, sizeof(double)* n * batches);
    hipMallocManaged(&b, sizeof(double)* m * batches);
    hipMallocManaged(&dx_aff, sizeof(double) * n *batches);
    hipMallocManaged(&dy_aff, sizeof(double) * m * batches);
    hipMallocManaged(&d_i1, sizeof(double) * m * batches);
    hipMallocManaged(&d_i2, sizeof(double) * m * batches);
    hipMallocManaged(&d_i3, sizeof(double) * m * batches);
    hipMallocManaged(&ds_aff, sizeof(double) * n * batches);
    hipMallocManaged(&d_i4, sizeof(double) * n * batches);
    hipMallocManaged(&rd, sizeof(double)* n * batches);
    hipMallocManaged(&rp, sizeof(double)* m * batches);
    hipMallocManaged(&rc, sizeof(double)* n * batches);
    hipMallocManaged(&v, sizeof(double)* n * batches);
    hipMallocManaged(&ap_aff, sizeof(double) * batches);
    hipMallocManaged(&ad_aff, sizeof(double) * batches);
    hipMallocManaged(&mu_aff, sizeof(double) * batches);
    hipMallocManaged(&mu, sizeof(double) * batches);
    hipMallocManaged(&mu_aff, sizeof(double) * batches);
    hipMallocManaged(&sigma, sizeof(double) * batches);
    hipMallocManaged(&ds_cor, sizeof(double) * n *batches);
    hipMallocManaged(&dy_cor, sizeof(double) * m * batches);
    hipMallocManaged(&perm, sizeof(int) * m * batches);
    // hipMallocManaged(&v, sizeof(double)* n * batches);
    hipMallocManaged(&cost, sizeof(double) * batches);
    CUDIE0();
    /*Initialization of the Problem*/
    double *ib, *ic, *ix, *is, *iy;
    ib = (double *)malloc(m * sizeof(double));
    ic = (double *)malloc(n * sizeof(double));
    ix = (double *)malloc(n * sizeof(double));
    is = (double *)malloc(n * sizeof(double));
    iy = (double *)malloc(m * sizeof(double));
    loadVec(ib, "Btxt.csv");
    loadVec(ic, "Ctxt.csv");
    loadVec(ix, "x_initial.csv");
    loadVec(is, "s_initial.csv");
    loadVec(iy, "y_initial.csv");
    //double ib[2] = {1, 2}, ic[4] = {-1, -1, 1, 1};
    //double ix[4] = {0.8, 0.1, 0.2, 1.9}, is[4] = {0.1, 0.2, 2.1, 2.2}, iy[2] = {-1.1, -1.2};
    
    double miny = 100, maxy = -100;
    for (int i = 0; i < batches; i++) {
        for (int j = 0; j < m; j++) {
            b[m*i + j] = ib[j];
            y[m*i + j] = iy[j];
            if (i == 0)
            {
                miny = min(miny, iy[j]);
                maxy = max(maxy, iy[j]);
            }
        }
    }  
    
    // cout << miny << " " << maxy << endl;      
    for (int i = 0; i < batches; i++) {
        for (int j = 0; j < n; j++) {
            c[n*i + j] = ic[j];
            x[n*i + j] = ix[j];
            s[n*i + j] = is[j];
        }
    }

    loadMat(csrValA, csrRowA, csrColA, "A_sparse.csv", &m, &nnz);
    loadMat(csrValAT, csrRowAT, csrColAT, "At_sparse.csv", &n, &nnz);
    /*printVector(csrColAT, 10);
    printVectordouble(csrValAT, 10);
    printVector(csrRowAT, 10);
    writeVecInt(csrColAT, nnz, "csrColAt_bef.txt");
    writeVecInt(csrRowAT, n+1, "csrRowAt_first.txt");*/
    //writeVec(csrValAT, 1000, "csrValAt_bef.txt");
    //sparesify(A, n, m, csrValAT, csrRowAT, csrColAT);
    int nnzAAt = nnz_aat, r_aat;
    cout << "Transpose Start" << endl;
    loadMat(csrValADA, csrRowADA, csrColADA, "AAt_sparse.csv", &r_aat, &nnzAAt);
    for (int i = 1; i < batches; i++)
    {
        //AAtransposeCPU(csrValA, csrRowA, csrColA, &csrValADA[nnzAAt*i],\
        //            &csrRowADA[(m+1)*i], &csrColADA[nnzAAt*i], m, n, nnz);
        for (int j = 0; j < nnzAAt; j++)
        {
            csrValADA[nnzAAt*i + j] = csrValADA[j];
        }
    }   
    cout << "Transpose End" << endl;
    CHECK_CUDA( hipMemcpy(csrValAAT, csrValADA,
                           nnz_aat * sizeof(double), hipMemcpyDefault) )
    CHECK_CUDA( hipMemcpy(csrColAAT, csrColADA,
                           nnz_aat * sizeof(int), hipMemcpyDefault) )
    CHECK_CUDA( hipMemcpy(csrRowAAT, csrRowADA,
                           (m + 1) * sizeof(int), hipMemcpyDefault) )
    /*cout << csrRowADA[1] << " " << csrRowADA[m] << endl;
    cout << csrValADA[1] <<  " " << csrValADA[nnzAAt] << endl;
    cout << csrColADA[1] <<  " " << csrColADA[nnzAAt+1] << endl; 
    cout << "Transpose End" << endl;*/
    CUDIE0();
    createA(A, csrValA, csrRowA, csrColA, m, n, nnz, base);
    /*printVectordouble(csrValADA, nnzAAt * batches);
    printVector(csrRowADA, (m + 1)*batches);
    printVector(csrColADA, nnzAAt * batches);*/


    // Handle creation
    hipsolverSpHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;
    hipsolverSpCreate(&cusolverH);
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipsolverSpSetStream(cusolverH, stream);

    //mat descriptor for AT
    hipsparseMatDescr_t descrAt = NULL;
    
    checkCudaErrors(hipsparseCreateMatDescr(&descrAt));
    checkCudaErrors(hipsparseSetMatType(descrAt, HIPSPARSE_MATRIX_TYPE_GENERAL));
    checkCudaErrors(hipsparseSetMatDiagType(descrAt, HIPSPARSE_DIAG_TYPE_NON_UNIT));
    checkCudaErrors(hipsparseSetMatIndexBase(descrAt, HIPSPARSE_INDEX_BASE_ZERO));
    
    /*---Initialization*/

    //printVectordouble(x, n*batches);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float ms = 0;
    hipEventRecord(start);

    /*Main Loop for IPM iterations*/
    for (int i = 0; i < iter; i++)
    {
        start_t = second();
        start_t = second();
        dot<<<batches, THREADS_PER_BLOCK>>>(x, s, mu, n, batches);
        hipDeviceSynchronize();
        CUDIE0();
        cout << "MU" << endl;
        for (int j = 0; j < min(batches, 10); j++)
            cout << mu[j]*n << " " << n << " ";
        cout << endl;
        //writeVecInt(csrRowAT, n+1, "csrRowAt_bef.txt");
        csr_mul_Av<<<batches, THREADS_PER_BLOCK>>>(csrValAT, csrRowAT, csrColAT, y, rd,\
                                                    n, m, nnz, batches, base);
        hipDeviceSynchronize();
        CUDIE0();
        /*cout << "xxxxxx" << endl;
        printVector(csrColAT, 10);
        printVectordouble(csrValAT, 10);
        printVector(csrRowAT, 10);
        printVectordouble(rd, 10);*/
        /*writeVecInt(csrColAT, 1000, "csrColAt.txt");
        writeVecInt(csrRowAT, n+1, "csrRowAt.txt");
        writeVec(csrValAT, 1000, "csrValAt.txt");*/
        //writeVec(rd, n, "yuss2.txt");
        
        vector_add<<<batches, THREADS_PER_BLOCK>>>(c, s, rd, rd, 1, -1, -1, n, batches);
        hipDeviceSynchronize();
        CUDIE0();
        // writeVec(rd, n, "yuss2.txt");
        
        csr_mul_Av<<<batches, THREADS_PER_BLOCK>>>(csrValA, csrRowA, csrColA, x, rp,\
                                                    m, n, nnz, batches, base);
        hipDeviceSynchronize();
        CUDIE0();
        vector_add<<<batches, THREADS_PER_BLOCK>>>(b, rp, rp, rp, 1, -1, 0, m, batches);
        hipDeviceSynchronize();
        CUDIE0();
        //writeVec(rp, m, "yuss1.txt");
        vector_mul<<<batches, THREADS_PER_BLOCK>>>(x, s, mu, rc, 0, -1, n, batches);
        hipDeviceSynchronize();
        CUDIE0();
        
        csr_diag_matmul<<<batches, THREADS_PER_BLOCK>>>(csrValA, csrRowA, csrColA,\
                                 csrValAS, s, s, 1, m, n, nnz, batches, base);
        hipDeviceSynchronize();
        CUDIE0();
        csr_mul_Av<<<batches, THREADS_PER_BLOCK>>>(csrValAS, csrRowA, csrColA, rc, d_i1,\
                                                    m, n, nnz, batches, base);
        hipDeviceSynchronize();
        CUDIE0();
        csr_diag_matmul<<<batches, THREADS_PER_BLOCK>>>(csrValA, csrRowA, csrColA,\
                                 csrValAD, x, s, 2, m, n, nnz, batches, base);
        hipDeviceSynchronize();
        CUDIE0();

        stop_t = second();
        time_preop = stop_t - start_t;

        cout << "ADAT start_t\n";
        start_t = second();
        start_t = second();
        
        ADAT<<<batches, THREADS_PER_BLOCK>>>(A, csrValAD, csrRowA, csrColA, csrValADA,\
                               csrRowADA, csrColADA, x, s, m, n, nnz, nnzAAt, batches, base);
        int crr = 0;
        cout << "ADAT start_t\n";
        // int ccmn(int *hA_csrOffsets, int *hA_columns, double *hA_values,int *hB_csrOffsets, int *hB_columns, double *hB_values, int *hC_csrOffsets, int *hC_columns, double *hC_values, int C_nnz, double *resC) 
        // crr = xsc(csrRowA, csrColA, csrValAD, csrRowAT, csrColAT, csrValAT, csrRowADA, csrColADA, csrValAAT, nnzAAt, csrValADA);
        //return 0;
        if (crr != 0)
        {
            printf("Error\n");
            return 0;
        }
        hipDeviceSynchronize();
        CUDIE0();
        stop_t = second();
        time_ada = stop_t - start_t;
        /*writeVec(csrValADA, nnzAAt, "csrValADA.txt");
        writeVecInt(csrColADA, nnzAAt, "csrColADA.txt");
        writeVecInt(csrRowADA, m + 1, "csrRowADA.txt");*/
        cout << "ADAT End\n";
        //return 0;
        csr_mul_Av<<<batches, THREADS_PER_BLOCK>>>(csrValAD, csrRowA, csrColA, rd, d_i2,\
                                                    m, n, nnz, batches, base);
        hipDeviceSynchronize();
        CUDIE0();
        vector_add<<<batches, THREADS_PER_BLOCK>>>(d_i1, d_i2, rp, d_i3, -1, 1, 1, m, batches);
        hipDeviceSynchronize();
        CUDIE0();
        
        //writeVec(d_i2, m, "yuss3.txt");
        //writeVec(d_i3, m, "yuss.txt");
        cout << m << " " << n << " " << nnzAAt<< " " << endl;
        //Solve for dy_aff
        solve(m, m, m, nnzAAt, batches, base, csrValADA, csrRowADA, csrColADA, dy_aff, d_i3, opts);

        start_t = second();
        start_t = second();
        csr_mul_Av<<<batches, THREADS_PER_BLOCK>>>(csrValAT, csrRowAT, csrColAT, dy_aff, ds_aff,\
                                                    n, m, nnz, batches, base);
        hipDeviceSynchronize();
        vector_add<<<batches, THREADS_PER_BLOCK>>>(ds_aff, rd, ds_aff, ds_aff, -1, 1, 0, n, batches);
        hipDeviceSynchronize();
        diag_vector_mul<<<batches, THREADS_PER_BLOCK>>>(s, s, rc, d_i4, n, 1, batches);
        hipDeviceSynchronize();
        diag_vector_mul<<<batches, THREADS_PER_BLOCK>>>(x, s, ds_aff, dx_aff, n, 0, batches);
        hipDeviceSynchronize();
        vector_add<<<batches, THREADS_PER_BLOCK>>>(dx_aff, d_i4, dx_aff, dx_aff, -1, 1, 0, n, batches);
        hipDeviceSynchronize();

        for (int j = 0; j < batches; j++)
        {
            ap_aff[j] = 1.0;
            ad_aff[j] = 1.0;
        }
        find_update_param<<<batches, THREADS_PER_BLOCK>>>(x, s, ap_aff, ad_aff, dx_aff,\
                                                ds_aff, n, batches);
        hipDeviceSynchronize();

        if (i > 35) {
            //Change from here
            findmuAff<<<batches, THREADS_PER_BLOCK>>>(x, s, mu_aff, ap_aff, dx_aff, ad_aff, ds_aff, n, batches);
            hipDeviceSynchronize();
            for (int j = 0; j < batches; j++)
            {
                sigma[j] = pow((mu_aff[j]/mu[j]), 3);
            }
            for (int j = 0; j < batches; j++)
            {
                mu_aff[j] = sigma[j] * mu[j];
            }
            vector_mul<<<batches, THREADS_PER_BLOCK>>>(dx_aff, ds_aff, mu_aff, v, 1, -1, n, batches);
            hipDeviceSynchronize();
            diag_vector_mul<<<batches, THREADS_PER_BLOCK>>>(x, x, v, ds_cor, n, 1, batches);
            hipDeviceSynchronize();
            cout << i << " LS Start\n";
            cusolverSpDcsrlsqvqrHost(cusolverH, n, m, nnz, descrAt, csrValAT, csrRowAT, csrColAT, ds_cor, 1e-10, &rankAt, dy_cor, perm, &min_norm);
            hipDeviceSynchronize();
            cout << i << " LS End\n";
            // for (int j = 0; j < m; j++)
            // {
            //     dy_cor[j] = -1.0 * dy_cor[j];
            // }
            vector_add<<<batches, THREADS_PER_BLOCK>>>(dy_aff, dy_cor, dy_cor, dy_aff, 1, -1, 0, m, batches);
            vector_add<<<batches, THREADS_PER_BLOCK>>>(ds_aff, ds_cor, ds_cor, ds_aff, 1, 1, 0, n, batches);
            hipDeviceSynchronize();
            //find corrector part
            find_update_param<<<batches, THREADS_PER_BLOCK>>>(x, s, ap_aff, ad_aff, dx_aff,\
                                                    ds_aff, n, batches);
            hipDeviceSynchronize();
        }
        update_vars<<<batches, THREADS_PER_BLOCK>>>(x, s, y, dx_aff, ds_aff, dy_aff, ap_aff,\
                                ad_aff, m, n, batches);
        hipDeviceSynchronize();








        //Till here
        // writeVec(x, n, "x_upd.txt");
        // writeVec(s, n, "s_upd.txt");
        cout << "Cost: " << endl;
        dot<<<batches, THREADS_PER_BLOCK>>>(c, x, cost, n, batches);
        hipDeviceSynchronize();
        stop_t = second();
        time_post = stop_t - start_t;
        for (int j = 0; j < min(batches, 10); j++)
            cout << cost[j]*n << " ";
        cout << endl;
        cout << "Preops time: " << time_preop << endl;
        cout << "ADA time: " << time_ada << endl;
        cout << "Postop_ts time: " << time_post << endl;
    }
    //for (int i  = 0; i < batches; i++)
      //  cout << mu[i] << endl;
    /*for (int i = 0; i < batches; i++){
        for (int j = 0; j < n; j++)
            cout << rd[i*n + j] << " ";
        cout << endl;
    }
    printVectordouble(csrValADA, 2*batches);
    printVectordouble(d_i1, m*batches);
    printVectordouble(d_i2, m*batches);
    printVectordouble(d_i3, m*batches);
    printVectordouble(x, n * batches);*/
    writeVec(x, n, "x_val.txt");
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf("Time taken CPU: %.4f\n", ms);

    hipFree(A);
    hipFree(csrValA);
    hipFree(csrRowA);
    hipFree(csrColA);
    hipFree(csrValAT);
    hipFree(csrRowAT);
    hipFree(csrColAT);
    hipFree(csrValAS);
    hipFree(csrValAD);
    hipFree(csrValADA);
    hipFree(csrRowADA);
    hipFree(csrColADA);
    hipFree(x);
    hipFree(s);
    hipFree(y);
    hipFree(c);
    hipFree(b);
    hipFree(dx_aff);
    hipFree(dy_aff);
    hipFree(d_i1);
    hipFree(d_i2);
    hipFree(d_i3);
    hipFree(ds_aff);
    hipFree(d_i4);
    hipFree(rd);
    hipFree(rp);
    hipFree(rc);
    hipFree(v);
    hipFree(ap_aff);
    hipFree(ad_aff);
    hipFree(mu_aff);
    hipFree(mu);
    hipFree(cost);
}
#define   A_NUM_ROWS 6590   // C compatibility
#define   C_NUM_NNZ 2042300   // C compatibility
int ccmn(int *hA_csrOffsets, int *hA_columns, double *hA_values,int *hB_csrOffsets, int *hB_columns, double *hB_values, int *hC_csrOffsets, int *hC_columns, double *hC_values, int C_nnz, double *resC) 
{
    printf("In\n"); fflush(stdout);
    //--------------------------------------------------------------------------
    //Device memory management: Allocate and copy A, B
    int   *dA_csrOffsets, *dA_columns, *dB_csrOffsets, *dB_columns,
          *dC_csrOffsets, *dC_columns;
    double *dA_values, *dB_values, *dC_values;
    hipsparseOperation_t opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseOperation_t opB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipDataType computeType = HIP_R_64F;
    double alpha = 1.0, beta = 0.0;
    int A_num_rows = 6590;
    int A_num_cols = 12879;
    int A_nnz = 483020;
    int batches = 16;
    int B_num_rows = A_num_cols;
    int B_num_cols = A_num_rows;
    int B_nnz = A_nnz;
    // allocate A
    printf("0\n");
    CHECK_CUDA( hipMalloc((void**) &dA_csrOffsets,
                           (A_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dA_columns, A_nnz * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dA_values,  A_nnz * sizeof(double)) )
    // allocate B
    CHECK_CUDA( hipMalloc((void**) &dB_csrOffsets,
                           (B_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dB_columns, B_nnz * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dB_values,  B_nnz * sizeof(double)) )
    // allocate C offsets
    CHECK_CUDA( hipMalloc((void**) &dC_csrOffsets,
                           (A_num_rows + 1) * sizeof(int)) )
    // copy A
    CHECK_CUDA( hipMemcpy(dA_csrOffsets, hA_csrOffsets,
                           (A_num_rows + 1) * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_columns, hA_columns, A_nnz * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_values, hA_values,
                           A_nnz * sizeof(double), hipMemcpyHostToDevice) )
    // copy B
    CHECK_CUDA( hipMemcpy(dB_csrOffsets, hB_csrOffsets,
                           (B_num_rows + 1) * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dB_columns, hB_columns, B_nnz * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dB_values, hB_values,
                           B_nnz * sizeof(double), hipMemcpyHostToDevice) )

    // //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA, matB, matC;
    void*  dBuffer1    = NULL;
    void*  dBuffer2    = NULL;
    void*  dBuffer3    = NULL;
    void*  dBuffer4    = NULL;
    void*  dBuffer5    = NULL;
    size_t bufferSize1 = 0;
    size_t bufferSize2 = 0;
    size_t bufferSize3 = 0;
    size_t bufferSize4 = 0;
    size_t bufferSize5 = 0;
    printf("1\n");
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_nnz,
                                      dA_csrOffsets, dA_columns, dA_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&matB, B_num_rows, B_num_cols, B_nnz,
                                      dB_csrOffsets, dB_columns, dB_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&matC, A_num_rows, B_num_cols, 0,
                                      NULL, NULL, NULL,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F) )
    //==========================================================================
    // SpGEMM Computation
    //==========================================================================
    printf("2\n");
    hipsparseSpGEMMDescr_t spgemmDesc;
    CHECK_CUSPARSE( hipsparseSpGEMM_createDescr(&spgemmDesc) )

    // ask bufferSize1 bytes for external memory
    CHECK_CUSPARSE(
        hipsparseSpGEMMreuse_workEstimation(handle, opA, opB, matA, matB, matC,
                                           HIPSPARSE_SPGEMM_DEFAULT,
                                           spgemmDesc, &bufferSize1, NULL)
    )
    CHECK_CUDA( hipMalloc((void**) &dBuffer1, bufferSize1) )
    // inspect the matrices A and B to understand the memory requirement for
    // the next step
    CHECK_CUSPARSE(
        hipsparseSpGEMMreuse_workEstimation(handle, opA, opB, matA, matB, matC,
                                           HIPSPARSE_SPGEMM_DEFAULT,
                                           spgemmDesc, &bufferSize1, dBuffer1)
    )
    // //--------------------------------------------------------------------------

    CHECK_CUSPARSE(
        hipsparseSpGEMMreuse_nnz(handle, opA, opB, matA, matB,
                                matC, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc,
                                &bufferSize2, NULL, &bufferSize3, NULL,
                                &bufferSize4, NULL)
    )
    CHECK_CUDA( hipMalloc((void**) &dBuffer2, bufferSize2) )
    CHECK_CUDA( hipMalloc((void**) &dBuffer3, bufferSize3) )
    CHECK_CUDA( hipMalloc((void**) &dBuffer4, bufferSize4) )
    CHECK_CUSPARSE(
        hipsparseSpGEMMreuse_nnz(handle, opA, opB, matA, matB,
                                matC, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc,
                                &bufferSize2, dBuffer2, &bufferSize3, dBuffer3,
                                &bufferSize4, dBuffer4)
    )
    CHECK_CUDA( hipFree(dBuffer1) )
    CHECK_CUDA( hipFree(dBuffer2) )
    //--------------------------------------------------------------------------
    printf("3\n");
    // get matrix C non-zero entries C_nnz1
    int64_t C_num_rows1, C_num_cols1, C_nnz1;
    CHECK_CUSPARSE( hipsparseSpMatGetSize(matC, &C_num_rows1, &C_num_cols1,
                                         &C_nnz1) )
    // allocate matrix C
    CHECK_CUDA( hipMalloc((void**) &dC_columns, C_nnz1 * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dC_values,  C_nnz1 * sizeof(double)) )
    CHECK_CUDA( hipMemset(dC_values, 0x0, C_nnz1 * sizeof(double)) )
    // fill dC_values if needed
    // update matC with the new pointers
    CHECK_CUSPARSE(
        hipsparseCsrSetPointers(matC, dC_csrOffsets, dC_columns, dC_values) )
    //--------------------------------------------------------------------------

    CHECK_CUSPARSE(
        hipsparseSpGEMMreuse_copy(handle, opA, opB, matA, matB, matC,
                                 HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc,
                                 &bufferSize5, NULL)
    )
    CHECK_CUDA( hipMalloc((void**) &dBuffer5, bufferSize5) )
    CHECK_CUSPARSE(
        hipsparseSpGEMMreuse_copy(handle, opA, opB, matA, matB, matC,
                                 HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc,
                                 &bufferSize5, dBuffer5)
    )
    CHECK_CUDA( hipFree(dBuffer3) )
    printf("4\n");
    // //--------------------------------------------------------------------------
  
    printf("5\n");
    for (int i = 0; i < batches; i++)
    {
        // update dA_values, dB_values
        CHECK_CUDA( hipMemcpy(dA_values, &hA_values[i * A_nnz],
                            A_nnz * sizeof(double), hipMemcpyDefault) )

        CHECK_CUDA( hipMemcpy(dB_values, hB_values,
                           B_nnz * sizeof(double), hipMemcpyHostToDevice) )
        CHECK_CUSPARSE(
        hipsparseSpGEMMreuse_compute(handle, opA, opB, &alpha, matA, matB, &beta,
                                    matC, computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                    spgemmDesc)
        )
        

        CHECK_CUDA( hipMemcpy(&resC[i * C_nnz], dC_values,
                            C_nnz * sizeof(double), hipMemcpyDefault) )
    }
    printf("6\n");
    //==========================================================================
    //==========================================================================
    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseSpGEMM_destroyDescr(spgemmDesc) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    // //--------------------------------------------------------------------------
    // // device result check
    // int   hC_csrOffsets_tmp[A_NUM_ROWS + 1];
    // int   hC_columns_tmp[C_NUM_NNZ];
    // double hC_values_tmp[C_NUM_NNZ];
    // CHECK_CUDA( hipMemcpy(hC_csrOffsets_tmp, dC_csrOffsets,
    //                        (A_num_rows + 1) * sizeof(int),
    //                        hipMemcpyDeviceToHost) )
    // CHECK_CUDA( hipMemcpy(hC_columns_tmp, dC_columns, C_nnz * sizeof(int),
    //                        hipMemcpyDeviceToHost) )
    // CHECK_CUDA( hipMemcpy(hC_values_tmp, dC_values, C_nnz * sizeof(double),
    //                        hipMemcpyDeviceToHost) )
    // int correct = 1;
    // printf("7\n");
    // for (int i = 0; i < A_num_rows + 1; i++) {
    //     if (hC_csrOffsets_tmp[i] != hC_csrOffsets[i]) {
    //         correct = 0;
    //         break;
    //     }
    // }
    // for (int i = 0; i < C_nnz; i++) {
    //     if (hC_columns_tmp[i] != hC_columns[i]) { // direct floating point
    //         correct = 0;                          // comparison is not reliable
    //         break;
    //     }
    // }
    // for (int i = 0; i < C_nnz; i++) {
    //     if (hC_columns_tmp[i] != hC_columns[i]) //||
    //         //hC_values_tmp[i]  != hC_values[i]) { // direct floating point
    //     {
    //         correct = 0;                         // comparison is not reliable
    //         break;
    //     }
    // }
    
    // if (correct)
    //     printf("spgemm_reuse_example test PASSED\n");
    // else {
    //     printf("spgemm_reuse_example test FAILED: wrong result\n");
    //     return EXIT_FAILURE;
    // }
    // //--------------------------------------------------------------------------
    // device memory deallocation
    CHECK_CUDA( hipFree(dBuffer4) )
    CHECK_CUDA( hipFree(dBuffer5) )
    CHECK_CUDA( hipFree(dA_csrOffsets) )
    CHECK_CUDA( hipFree(dA_columns) )
    CHECK_CUDA( hipFree(dA_values) )
    CHECK_CUDA( hipFree(dB_csrOffsets) )
    CHECK_CUDA( hipFree(dB_columns) )
    CHECK_CUDA( hipFree(dB_values) )
    CHECK_CUDA( hipFree(dC_csrOffsets) )
    CHECK_CUDA( hipFree(dC_columns) )
    CHECK_CUDA( hipFree(dC_values) )
    return 0;
}

int xsc(int *hA_csrOffsets, int *hA_columns, double *hA_values,int *hB_csrOffsets, int *hB_columns, double *hB_values, int *hC_csrOffsets, int *hC_columns, double *hC_values, int C_nnz, double *resC) 
{
    printf("HERE\n");
    return ccmn(hA_csrOffsets,hA_columns,hA_values,hB_csrOffsets, hB_columns, hB_values,hC_csrOffsets, hC_columns, hC_values, C_nnz, resC);
}
void printVectordouble(const double *V, int m)
{
    for (int i = 0; i < m; i++)
        std::cout << V[i] << " ";
    std::cout << std::endl;
}
// Generate the three vectors A, IA, JA 
// void sparesify(double *M, int m, int n, double *A, int *IA, int *JA)
// {
//     //int m = M.size(), n = M[0].size();
//     int i, j;
//     //vi A;
//     IA[0] = 1; // IA matrix has N+1 rows
//     //vi JA;
//     int NNZ = 0;
  
//     for (i = 0; i < m; i++) {
//         for (j = 0; j < n; j++) {
//             if (M[i + m*j] != 0) {
//                 A[NNZ] = M[i + m*j];
//                 JA[NNZ] = j + 1;
  
//                 // Count Number of Non Zero 
//                 // Elements in row i
//                 NNZ++;
//             }
//         }
//         IA[i + 1] = NNZ + 1;
//     }
  
//     printMatrix(M, m, n);
//     printVectordouble(A, NNZ);
//     printVector(IA, m + 1);
//     printVector(JA, NNZ);
// }
//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
 void printMatrix(const double *A, int nr_rows_A, int nr_cols_A) {
 
     for(int i = 0; i < nr_rows_A; ++i){
         for(int j = 0; j < nr_cols_A; ++j){
             std::cout << A[j * nr_rows_A + i] << " ";
         }
         std::cout << std::endl;
     }
     std::cout << std::endl;
 }
  
// Utility Function to print A, IA, JA vectors
// with some decoration.
void printVector(const int *V, int m)
{
    for (int i = 0; i < m; i++)
        std::cout << V[i] << " ";
    std::cout << std::endl;
}

/*
* Find Number of Non Zero Elements in a Sparse Matrix
*/
int findNNZ(const double *M, int N)
{
    int nnz = 0;
    for (int i = 0;  i < N; i++)
    {
        if (M[i] != 0.0)
            nnz++;
    }

    return nnz;
}

//Get matrix dimensions of CSR Matrix
void getMatSize(string fname, int *rows, int *nnz)
{
    ifstream infile;
    infile.open(fname);
    string line;
    getline(infile, line);
    int i, n = 0;
    stringstream stream(line);
    
    while (stream >> i)
        n++;
    *rows = n - 1;
    
    getline(infile, line);
    n = 0;
    stringstream stream1(line);
    while (stream1 >> i)
        n++;
    *nnz = n;
}

// Load CSR matrix
void loadMat(double *A, int *IA, int *JA, string fname, int *rows, int *nnz)
{
    ifstream infile;
    infile.open(fname);
    string line;
    getline(infile, line);
    int i, n = 0;
    //cout << line << endl;
    stringstream stream(line);
    while (stream >> i)
    {
        IA[n] = i-1;
        n++;
    }
    *rows = n - 1;
    getline(infile, line);
    n = 0;
    stringstream stream1(line);
    while (stream1 >> i)
    {
        JA[n] = i-1;
        n++;
        //if (n < 50)
            //cout << i << " ";
    }
    cout << endl;
    *nnz = n;
    getline(infile, line);
    n = 0;
    double f;
    stringstream stream2(line);
    while (stream2 >> f)
    {
        A[n] = f;
        n++;
        //if (n < 50)
            //cout << f << " ";
    }
    cout << *rows << " " << *nnz << " " << n << endl;
}

// Load Vector
void loadVec(double *V, string fname)
{
    int cols = 0, rows = 0;
    ifstream infile;
    infile.open(fname);
    while(!infile.eof())
    {
        string line;
        getline(infile, line);
        int temp_cols = 0;
        stringstream stream(line);
        double f;
        while(stream >> f)
        {
            V[cols*rows + temp_cols++] = f;
        }
        
        if(temp_cols == 0)
            continue;
        
        if (cols == 0)
            cols = temp_cols;
        
        rows++;
    }
    cout << rows << " " << cols << endl;
    infile.close();
}

void writeVec(double *V, int n, string fname)
{
    ofstream outdata;
    outdata.open(fname);
    for (int i = 0; i < n; i++)
    {
        outdata << V[i] << endl;
    }
    outdata.close();
}
void writeVecInt(int *V, int n, string fname)
{
    ofstream outdata;
    outdata.open(fname);
    for (int i = 0; i < n; i++)
    {
        outdata << V[i] << endl;
    }
    outdata.close();
}

void createA(double *A, double *csrValA, int *csrRowA, int *csrColA, int m, int n, int nnz, int baseA)
{
    for (int i  = 0; i < m; i++)
        for (int j = 0; j < n; j++)
            A[i + m*j] = 0;

    for (int i = 0; i < m; i++)
    {
        for (int j = csrRowA[i] - baseA; j < csrRowA[i+1] - baseA; j++)
        {
            A[i + m*(csrColA[j]-baseA)] = csrValA[j];
        }
    }
}


/*
 * Copyright 2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 *  A framework of refactorization process.
 *  
 *  step 1: compute P*A*Q = L*U by
 *    - reordering and
 *    - LU with partial pivoting in cusolverSp
 *
 *  step 2: set up cusolverRf by (P, Q, L, U)
 *
 *  step 3: analyze and refactor A
 *
 *  How to use
 *     ./cuSolverRf -P=symrcm -file <file>
 *     ./cuSolverRf -P=symamd -file <file>
 *
 */



// template <typename T_ELEM>
// int loadMMSparseMatrix(
//     char *filename,
//     char elem_type,
//     bool csrFormat,
//     int *m,
//     int *n,
//     int *nnz,
//     T_ELEM **aVal,
//     int **aRowInd,
//     int **aColInd,
//     int extendSymMatrix);

void UsageRF(void)
{
    printf( "<options>\n");
    printf( "-h          : display this help\n");
    printf( "-P=<name>    : choose a reordering\n");
    printf( "              symrcm (Reverse Cuthill-McKee)\n");
    printf( "              symamd (Approximate Minimum Degree)\n");
    printf( "-file=<filename> : filename containing a matrix in MM format\n");
    printf( "-bs=<batch_size> : normally 32 - 128, default=32 \n");
    printf( "-device=<device_id> : <device_id> if want to run on specific GPU\n");

    exit( 0 );
}
/* compute | b - A*x|_inf */
void residaul_eval(
    int n,
    const hipsparseMatDescr_t descrA,
    const double *csrVal,
    const int *csrRowPtr,
    const int *csrColInd,
    const double *b,
    const double *x,
    double *r_nrminf_ptr)
{
    const int base = (hipsparseGetMatIndexBase(descrA) != HIPSPARSE_INDEX_BASE_ONE)? 0:1 ;
    const int lower = (HIPSPARSE_FILL_MODE_LOWER == hipsparseGetMatFillMode(descrA))? 1:0;
    const int unit  = (HIPSPARSE_DIAG_TYPE_UNIT == hipsparseGetMatDiagType(descrA))? 1:0;

    double r_nrminf = 0;
    for(int row = 0 ; row < n ; row++){
        const int start = csrRowPtr[row]   - base;
        const int end   = csrRowPtr[row+1] - base;
        double dot = 0;
        for(int colidx = start ; colidx < end; colidx++){
            const int col = csrColInd[colidx] - base;
            double Aij = csrVal[colidx];
            double xj  = x[col];
            if ( (row == col) && unit ){
                Aij = 1.0;
            }
            int valid = (row >= col) && lower ||
                        (row <= col) && !lower ;
            if ( valid ){
                dot += Aij*xj;
            }
        }
        double ri = b[row] - dot;
        r_nrminf = (r_nrminf > fabs(ri))? r_nrminf : fabs(ri);
    }
    *r_nrminf_ptr = r_nrminf;
}
void parseCommandLineArguments(int argc, char *argv[], struct testOpts &opts)
{
    memset(&opts, 0, sizeof(opts));

    if (checkCmdLineFlag(argc, (const char **)argv, "-h"))
    {
        UsageRF();
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "P"))
    {
        char *reorderType = NULL;
        getCmdLineArgumentString(argc, (const char **)argv, "P", &reorderType);

        if (reorderType)
        {
            if ((STRCASECMP(reorderType, "symrcm") != 0) && (STRCASECMP(reorderType, "symamd") != 0))
            {
                printf("\nIncorrect argument passed to -P option\n");
                UsageRF();
            }
            else
            {
                opts.reorder = reorderType;
            }
        }
    }

    if (!opts.reorder)
    {
        opts.reorder = "symrcm"; // Setting default reordering to be symrcm.
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "file"))
    {
        char *fileName = 0;
        getCmdLineArgumentString(argc, (const char **)argv, "file", &fileName);

        if (fileName)
        {
            opts.sparse_mat_filename = fileName;
        }
        else
        {
            printf("\nIncorrect filename passed to -file \n ");
            UsageRF();
        }
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "bs"))
    {
        char *batch_size = 0;
        getCmdLineArgumentString(argc, (const char **)argv, "bs", &batch_size);

        if (batch_size)
        {
            opts.batch_size = atoi(batch_size);
        }
        else
        {
            printf("\nIncorrect batch size passed to -bs \n ");
            UsageRF();
        }
    }
}

int solve (int rowsA, int colsA, int N, int nnzA, int batchSize, int baseA, \
            double *h_csrValA, int *h_csrRowPtrA, int *h_csrColIndA, \
             double *x, double *b, struct testOpts opts)
{
    //struct testOpts opts;
    hipsolverRfHandle_t cusolverRfH = NULL; // refactorization
    hipsolverSpHandle_t cusolverSpH = NULL; // reordering, permutation and 1st LU factorization
    hipsparseHandle_t   cusparseH = NULL;   // residual evaluation
    hipStream_t stream = NULL;
    hipsparseMatDescr_t descrA = NULL; // A is a base-0 general matrix

    csrluInfoHost_t info = NULL; // opaque info structure for LU with parital pivoting

    /*int batchSize = 32;
    int rowsA = 0; // number of rows of A
    int colsA = 0; // number of columns of A
    int N     = 0; // n = rowsA = colsA
    int nnzA  = 0; // number of nonzeros of A
    int baseA = 0; // base index in CSR format
                   // cusolverRf only works for base-0*/

// cusolverRf only works for square matrix, 
// assume n = rowsA = colsA

    // CSR(A) from I/O
    //int *h_csrRowPtrA = NULL; // <int> n+1 
    //int *h_csrColIndA = NULL; // <int> nnzA 
    //double *h_csrValA = NULL; // <double> nnzA 
    //array of pointers to the values of each matrix in the batch (of size
    //batchSize) on the host
    double **h_A_array = NULL;
    //For example, if h_A_batch is the array (of size batchSize*nnzA) containing 
    //the values of each matrix in the batch written contiguosly one matrix  
    //after another on the host, then h_A_array[j] = &h_A_batch[nnzA*j];
    //for j=0,...,batchSize-1.
    double *h_A_batch=NULL; 

    int *h_Qreorder = NULL; // <int> n
                            // reorder to reduce zero fill-in
                            // Qreorder = symrcm(A) or Qreroder = symamd(A)
    // B = Q*A*Q^T
    int *h_csrRowPtrB = NULL; // <int> n+1
    int *h_csrColIndB = NULL; // <int> nnzA
    double *h_csrValB = NULL; // <double> nnzA
    int *h_mapBfromA = NULL;  // <int> nnzA

    double *h_x = NULL; // <double> n,  x = A \ b
    double *h_b = NULL; // <double> n, b = ones(m,1)
    double *h_r = NULL; // <double> n, r = b - A*x
    //array (of size batchSize*n*nrhs) containing the values of each rhs in 
    //the batch written contiguously one rhs after another on the host
    //nrhs is # of rhs for each system (currently only =1 is supported) 
    double *h_X_batch = NULL;
    double **h_X_array = NULL;

    // solve B*(Qx) = Q*b
    double *h_xhat = NULL; // <double> n, Q*x_hat = x
    double *h_bhat = NULL; // <double> n, b_hat = Q*b 

    size_t size_perm = 0;
    size_t size_internal = 0; 
    size_t size_lu  = 0; // size of working space for csrlu
    void *buffer_cpu = NULL; // working space for
                             // - permutation: B = Q*A*Q^T
                             // - LU with partial pivoting in cusolverSp

    // cusolverSp computes LU with partial pivoting
    //     Plu*B*Qlu^T = L*U
    //   where B = Q*A*Q^T
    //
    // nnzL and nnzU are not known until factorization is done.
    // However upper bound of L+U is known after symbolic analysis of LU.
    int *h_Plu = NULL; // <int> n
    int *h_Qlu = NULL; // <int> n

    int nnzL = 0;
    int *h_csrRowPtrL = NULL; // <int> n+1
    int *h_csrColIndL = NULL; // <int> nnzL
    double *h_csrValL = NULL; // <double> nnzL

    int nnzU = 0;
    int *h_csrRowPtrU = NULL; // <int> n+1
    int *h_csrColIndU = NULL; // <int> nnzU
    double *h_csrValU = NULL; // <double> nnzU

    int *h_P = NULL; // <int> n, P = Plu * Qreorder
    int *h_Q = NULL; // <int> n, Q = Qlu * Qreorder

    int *d_csrRowPtrA = NULL; // <int> n+1
    int *d_csrColIndA = NULL; // <int> nnzA
    double *d_csrValA = NULL; // <double> nnzA
    
    //array of pointers to the values of each matrix in the batch (of size
    //batchSize) on the device
    double **d_A_array=NULL;
    //For example, if d_A_batch is the array (of size batchSize*nnzA) containing 
    //the values of each matrix in the batch written contiguosly one matrix  
    //after another on the device, then d_A_array[j] = &d_A_batch[nnzA*j];
    //for j=0,...,batchSize-1.
    double *d_A_batch=NULL; 

    double *d_x = NULL; // <double> n, x = A \ b 
    double *d_b = NULL; // <double> n, a copy of h_b
    double *d_r = NULL; // <double> n, r = b - A*x

    //array (of size batchSize*n*nrhs) containing the values of each rhs in 
    //the batch written contiguously one rhs after another on the device
    double *d_X_batch = NULL;
    double **d_X_array = NULL;

    int *d_P = NULL; // <int> n, P*A*Q^T = L*U
    int *d_Q = NULL; // <int> n 
  
    double *d_T = NULL; // working space in hipsolverRfSolve
                        // |d_T| = 2*batchSize*n*nrhs

    // the constants used in residual evaluation, r = b - A*x
    const double minus_one = -1.0;
    const double one = 1.0;
    // the constants used in cusolverRf
    // nzero is the value below which zero pivot is flagged.
    // nboost is the value which is substitured for zero pivot.
    double nzero = 0.0;
    double nboost= 0.0;
    // the constant used in cusolverSp
    // singularity is -1 if A is invertible under tol
    // tol determines the condition of singularity
    // pivot_threshold decides pivoting strategy            
    int singularity = 0; 
    const double tol = 1.e-14;
    const double pivot_threshold = 1.0;
    // the constants used in cusolverRf
    const hipsolverRfFactorization_t fact_alg = HIPSOLVERRF_FACTORIZATION_ALG0; // default
    const hipsolverRfTriangularSolve_t solve_alg = HIPSOLVERRF_TRIANGULAR_SOLVE_ALG1; // default

    double x_inf = 0.0; // |x|
    double r_inf = 0.0; // |r|
    double A_inf = 0.0; // |A|
    //int errors = 0;

    double start, stop;
    double time_reorder;
    double time_perm;
    double time_sp_analysis;
    double time_sp_factor;
    double time_sp_solve;
    double time_sp_extract;
    double time_rf_assemble;
    double time_rf_reset;
    double time_rf_refactor;
    double time_rf_solve;

    printf("step 1.1: preparation\n");
    printf("step 1.1: read matrix market format\n");

    if ( rowsA != colsA )
    {
        fprintf(stderr, "Error: only support square matrix\n");
        return 1;
    }

    printf("WARNING: cusolverRf only works for base-0 \n");
    if (baseA)
    {
        for(int i = 0 ; i <= rowsA ; i++)
        {
            h_csrRowPtrA[i]--;
        }
        for(int i = 0 ; i < nnzA ; i++)
        {
            h_csrColIndA[i]--;
        }
        baseA = 0;
    }

    N = rowsA;
    printf("sparse matrix A is %d x %d with %d nonzeros, base=%d\n", rowsA, colsA, nnzA, baseA);

    // setup batch of A
    h_A_array = (double**)malloc(sizeof(double*)*batchSize);
    h_A_batch = (double*)malloc(sizeof(double)*batchSize*nnzA);
    for (int i = 0; i < batchSize; ++i)
    {
        memcpy(&h_A_batch[i*nnzA], &h_csrValA[i*nnzA], sizeof(double)*nnzA);
    }

    checkCudaErrors(hipsolverSpCreate(&cusolverSpH));
    checkCudaErrors(hipsparseCreate(&cusparseH));
    checkCudaErrors(hipStreamCreate(&stream));

    checkCudaErrors(hipsolverSpSetStream(cusolverSpH, stream));
    checkCudaErrors(hipsparseSetStream(cusparseH, stream));

    checkCudaErrors(hipsparseCreateMatDescr(&descrA));
    checkCudaErrors(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));

    if (baseA) 
    {
        checkCudaErrors(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE));
    }
    else
    {
        checkCudaErrors(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));
    }

    h_Qreorder   = (int*)malloc(sizeof(int)*colsA);

    h_csrRowPtrB = (int*   )malloc(sizeof(int)*(rowsA+1));
    h_csrColIndB = (int*   )malloc(sizeof(int)*nnzA);
    h_csrValB    = (double*)malloc(sizeof(double)*nnzA);
    h_mapBfromA  = (int*   )malloc(sizeof(int)*nnzA);

    h_x    = (double*)malloc(sizeof(double)*colsA);
    h_X_array = (double**)malloc(sizeof(double*)*batchSize);
    h_X_batch = (double*)malloc(sizeof(double)*batchSize*N);
    h_b    = (double*)malloc(sizeof(double)*rowsA);
    h_r    = (double*)malloc(sizeof(double)*rowsA);
    h_xhat = (double*)malloc(sizeof(double)*colsA);
    h_bhat = (double*)malloc(sizeof(double)*rowsA);

    assert(NULL != h_Qreorder);

    assert(NULL != h_csrRowPtrB);
    assert(NULL != h_csrColIndB);
    assert(NULL != h_csrValB   );
    assert(NULL != h_mapBfromA);

    assert(NULL != h_x);
    assert(NULL != h_b);
    assert(NULL != h_r);
    assert(NULL != h_xhat);
    assert(NULL != h_bhat);

    checkCudaErrors(hipMalloc((void **)&d_csrRowPtrA, sizeof(int)*(rowsA+1)));
    checkCudaErrors(hipMalloc((void **)&d_csrColIndA, sizeof(int)*nnzA));
    checkCudaErrors(hipMalloc((void **)&d_csrValA   , sizeof(double)*nnzA));
    checkCudaErrors(hipMalloc((void **)&d_A_array   , sizeof(double*)*batchSize));
    checkCudaErrors(hipMalloc((void **)&d_A_batch   , sizeof(double)*batchSize*nnzA));
    for (int i = 0; i < batchSize; ++i)
    {
        h_A_array[i] = &(d_A_batch[i*nnzA]);
    }
    checkCudaErrors(hipMemcpy(d_A_array, h_A_array, batchSize * sizeof(double*), hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void **)&d_x, sizeof(double)*colsA));
    checkCudaErrors(hipMalloc((void **)&d_X_array, sizeof(double*)*batchSize));
    checkCudaErrors(hipMalloc((void **)&d_X_batch, sizeof(double)*batchSize*N));
    for (int i = 0; i < batchSize; ++i)
    {
        h_X_array[i] = &(d_X_batch[i*rowsA]);
    }
    checkCudaErrors(hipMemcpy(d_X_array, h_X_array, batchSize * sizeof(double*), hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void **)&d_b, sizeof(double)*rowsA));
    checkCudaErrors(hipMalloc((void **)&d_r, sizeof(double)*rowsA));
    checkCudaErrors(hipMalloc((void **)&d_P, sizeof(int)*rowsA));
    checkCudaErrors(hipMalloc((void **)&d_Q, sizeof(int)*colsA));
    checkCudaErrors(hipMalloc((void **)&d_T, sizeof(double)*rowsA*2*batchSize));



    /***/
    
    
    /***/
    printf("step 1.2: set random right hand side vector (b) in range -1 to 1\n");
    for(int row = 0 ; row < rowsA ; row++){
        h_b[row] = b[row];
    }
    srand(time(NULL));
    for(int i = 0; i < batchSize*colsA; ++i)
    {
        h_X_batch[i] = b[i];//(double)rand()/RAND_MAX*2.0-1.0; + 0.0001 * i/colsA
    }
    
    printf("step 2: reorder the matrix to reduce zero fill-in\n");
    printf("        Q = symrcm(A) or Q = symamd(A) \n");
    start = second();
    start = second();

    if ( 0 == strcmp(opts.reorder, "symrcm") )
    {
        checkCudaErrors(cusolverSpXcsrsymrcmHost(
            cusolverSpH, rowsA, nnzA,
            descrA, h_csrRowPtrA, h_csrColIndA, 
            h_Qreorder));
    }
    else if ( 0 == strcmp(opts.reorder, "symamd") )
    {
        checkCudaErrors(cusolverSpXcsrsymamdHost(
            cusolverSpH, rowsA, nnzA,
            descrA, h_csrRowPtrA, h_csrColIndA, 
            h_Qreorder));
    }
    else 
    {
        fprintf(stderr, "Error: %s is unknow reordering\n", opts.reorder);
        return 1;
    }

    stop = second();
    time_reorder = stop - start;

    printf("step 3: B = Q*A*Q^T\n");
    memcpy(h_csrRowPtrB, h_csrRowPtrA, sizeof(int)*(rowsA+1));
    memcpy(h_csrColIndB, h_csrColIndA, sizeof(int)*nnzA);
    
    start = second();
    start = second();

    checkCudaErrors(cusolverSpXcsrperm_bufferSizeHost(
        cusolverSpH, rowsA, colsA, nnzA,
        descrA, h_csrRowPtrB, h_csrColIndB,
        h_Qreorder, h_Qreorder,
        &size_perm));

    if (buffer_cpu) {
        free(buffer_cpu); 
    }
    buffer_cpu = (void*)malloc(sizeof(char)*size_perm);
    assert(NULL != buffer_cpu);

    // h_mapBfromA = Identity 
    for(int j = 0 ; j < nnzA ; j++){
        h_mapBfromA[j] = j;
    }
    checkCudaErrors(cusolverSpXcsrpermHost(
        cusolverSpH, rowsA, colsA, nnzA,
        descrA, h_csrRowPtrB, h_csrColIndB,
        h_Qreorder, h_Qreorder,
        h_mapBfromA,
        buffer_cpu));

    // B = A( mapBfromA )
    for(int j = 0 ; j < nnzA ; j++){
        h_csrValB[j] = h_csrValA[ h_mapBfromA[j] ];
    }

    stop = second();
    time_perm = stop - start;

    printf("step 4: solve A*x = b by LU(B) in cusolverSp\n");

    printf("step 4.1: create opaque info structure\n");
    checkCudaErrors(cusolverSpCreateCsrluInfoHost(&info));

    printf("step 4.2: analyze LU(B) to know structure of Q and R, and upper bound for nnz(L+U)\n");
    start = second();
    start = second();

    checkCudaErrors(cusolverSpXcsrluAnalysisHost(
        cusolverSpH, rowsA, nnzA,
        descrA, h_csrRowPtrB, h_csrColIndB,
        info));

    stop = second();
    time_sp_analysis = stop - start;

    printf("step 4.3: workspace for LU(B)\n");
    checkCudaErrors(cusolverSpDcsrluBufferInfoHost(
        cusolverSpH, rowsA, nnzA,
        descrA, h_csrValB, h_csrRowPtrB, h_csrColIndB,
        info,
        &size_internal,
        &size_lu));

    if (buffer_cpu) { 
        free(buffer_cpu); 
    }
    buffer_cpu = (void*)malloc(sizeof(char)*size_lu);
    assert(NULL != buffer_cpu);

    printf("step 4.4: compute Ppivot*B = L*U \n");
    start = second();
    start = second();

    checkCudaErrors(cusolverSpDcsrluFactorHost(
        cusolverSpH, rowsA, nnzA,
        descrA, h_csrValB, h_csrRowPtrB, h_csrColIndB,
        info, pivot_threshold,
        buffer_cpu));

    stop = second();
    time_sp_factor = stop - start;

    // TODO: check singularity by tol
    printf("step 4.5: check if the matrix is singular \n");
    checkCudaErrors(cusolverSpDcsrluZeroPivotHost(
        cusolverSpH, info, tol, &singularity));

    if ( 0 <= singularity){
        fprintf(stderr, "Error: A is not invertible, singularity=%d\n", singularity);
        return 1;
    }


    printf("step 4.6: solve A*x = b \n");
    printf("    i.e.  solve B*(Qx) = Q*b \n");
    start = second();
    start = second();

    // b_hat = Q*b
    for(int j = 0 ; j < rowsA ; j++){
        h_bhat[j] = h_b[h_Qreorder[j]];
    }
    // B*x_hat = b_hat
    checkCudaErrors(cusolverSpDcsrluSolveHost(
        cusolverSpH, rowsA, h_bhat, h_xhat, info, buffer_cpu));

    // x = Q^T * x_hat
    for(int j = 0 ; j < rowsA ; j++){
        h_x[h_Qreorder[j]] = h_xhat[j];
    }

    stop = second();
    time_sp_solve = stop - start;

    printf("step 4.7: evaluate residual r = b - A*x (result on CPU)\n");
    // use GPU gemv to compute r = b - A*x
    checkCudaErrors(hipMemcpy(d_csrRowPtrA, h_csrRowPtrA, sizeof(int)*(rowsA+1), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_csrColIndA, h_csrColIndA, sizeof(int)*nnzA     , hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_csrValA   , h_csrValA   , sizeof(double)*nnzA  , hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(d_r, h_b, sizeof(double)*rowsA, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_x, h_x, sizeof(double)*colsA, hipMemcpyHostToDevice));
    cout << "h_x" << endl;
    //for  (int i = 0; i < colsA; i++)
    //    cout << h_x[i] << " ";
    //cout << endl;
    /* Wrap raw data into cuSPARSE generic API objects */
    hipsparseSpMatDescr_t matA = NULL;
    if (baseA)
    {
        checkCudaErrors(hipsparseCreateCsr(&matA, rowsA, colsA, nnzA, d_csrRowPtrA, d_csrColIndA, d_csrValA,
                                        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                        HIPSPARSE_INDEX_BASE_ONE, HIP_R_64F));
    }
    else
    {
        checkCudaErrors(hipsparseCreateCsr(&matA, rowsA, colsA, nnzA, d_csrRowPtrA, d_csrColIndA, d_csrValA,
                                        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));
    }
    //hipsparseDcsrmv HIPSPARSE_OPERATION_NON_TRANSPOSE
    /*checkCudaErrors(hipsparseDbsrmv(cusparseH,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        rowsA,
        colsA,
        nnzA,
        &minus_one,
        descrA,
        d_csrValA,
        d_csrRowPtrA,
        d_csrColIndA,
        d_x,
        &one,
        d_r));*/
    hipsparseDnVecDescr_t vecx = NULL;
    checkCudaErrors(hipsparseCreateDnVec(&vecx, colsA, d_x, HIP_R_64F));
    hipsparseDnVecDescr_t vecAx = NULL;
    checkCudaErrors(hipsparseCreateDnVec(&vecAx, rowsA, d_r, HIP_R_64F));
    size_t bufferSize = 0;
    checkCudaErrors(hipsparseSpMV_bufferSize(
        cusparseH, HIPSPARSE_OPERATION_NON_TRANSPOSE, &minus_one, matA, vecx,
        &one, vecAx, HIP_R_64F, HIPSPARSE_MV_ALG_DEFAULT, &bufferSize));
    void *buffer = NULL;
    cout << "Buffer Size " << bufferSize*1000 << endl;
    checkCudaErrors(hipMalloc(&buffer, bufferSize));
    checkCudaErrors(hipsparseSpMV(cusparseH,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &minus_one,
            matA,
            vecx,
            &one,
            vecAx, HIP_R_64F,HIPSPARSE_MV_ALG_DEFAULT, buffer));
    checkCudaErrors(hipMemcpy(h_r, d_r, sizeof(double)*rowsA, hipMemcpyDeviceToHost));
    /*cout << "h_r" << endl;
    for  (int i = 0; i < rowsA; i++)
        cout << h_r[i] << " ";
    cout << endl;*/
    x_inf = vec_norminf(colsA, h_x);
    r_inf = vec_norminf(rowsA, h_r);
    A_inf = csr_mat_norminf(rowsA, colsA, nnzA, descrA, h_csrValA, h_csrRowPtrA, h_csrColIndA);

    printf("(CPU) |b - A*x| = %E \n", r_inf);
    printf("(CPU) |A| = %E \n", A_inf);
    printf("(CPU) |x| = %E \n", x_inf);
    printf("(CPU) |b - A*x|/(|A|*|x|) = %E \n", r_inf/(A_inf * x_inf));

    printf("step 5: extract P, Q, L and U from P*B*Q^T = L*U \n");
    printf("        L has implicit unit diagonal\n");
    start = second();
    start = second();

    checkCudaErrors(cusolverSpXcsrluNnzHost(
        cusolverSpH,
        &nnzL,
        &nnzU,
        info));

    h_Plu = (int*)malloc(sizeof(int)*rowsA);
    h_Qlu = (int*)malloc(sizeof(int)*colsA);

    h_csrValL    = (double*)malloc(sizeof(double)*nnzL);
    h_csrRowPtrL = (int*)malloc(sizeof(int)*(rowsA+1)); 
    h_csrColIndL = (int*)malloc(sizeof(int)*nnzL);

    h_csrValU    = (double*)malloc(sizeof(double)*nnzU);
    h_csrRowPtrU = (int*)malloc(sizeof(int)*(rowsA+1)); 
    h_csrColIndU = (int*)malloc(sizeof(int)*nnzU);

    assert(NULL != h_Plu);
    assert(NULL != h_Qlu);

    assert(NULL != h_csrValL);
    assert(NULL != h_csrRowPtrL);
    assert(NULL != h_csrColIndL);

    assert(NULL != h_csrValU);
    assert(NULL != h_csrRowPtrU);
    assert(NULL != h_csrColIndU);

    checkCudaErrors(cusolverSpDcsrluExtractHost(
        cusolverSpH,
        h_Plu,
        h_Qlu,
        descrA,
        h_csrValL, 
        h_csrRowPtrL,
        h_csrColIndL,
        descrA,
        h_csrValU,
        h_csrRowPtrU,
        h_csrColIndU,
        info,
        buffer_cpu));

    stop = second();
    time_sp_extract = stop - start;

    printf("nnzL = %d, nnzU = %d\n", nnzL, nnzU);

/*  B = Qreorder*A*Qreorder^T
 *  Plu*B*Qlu^T = L*U
 *
 *  (Plu*Qreorder)*A*(Qlu*Qreorder)^T = L*U
 *  
 *  Let P = Plu*Qreroder, Q = Qlu*Qreorder, 
 *  then we have
 *      P*A*Q^T = L*U
 *  which is the fundamental relation in cusolverRf.
 */
    printf("step 6: form P*A*Q^T = L*U\n");

    h_P = (int*)malloc(sizeof(int)*rowsA);
    h_Q = (int*)malloc(sizeof(int)*colsA);
    assert(NULL != h_P);
    assert(NULL != h_Q);

    printf("step 6.1: P = Plu*Qreroder\n");
    // gather operation, P = Qreorder(Plu)
    for(int j = 0 ; j < rowsA ; j++){
        h_P[j] = h_Qreorder[h_Plu[j]];
    }

    printf("step 6.2: Q = Qlu*Qreorder \n");
    // gather operation, Q = Qreorder(Qlu)
    for(int j = 0 ; j < colsA ; j++){
        h_Q[j] = h_Qreorder[h_Qlu[j]];
    }

    printf("step 7: create cusolverRf handle\n");
    checkCudaErrors(hipsolverRfCreate(&cusolverRfH));

    printf("step 8: set parameters for cusolverRf \n");
    // numerical values for checking "zeros" and for boosting.
    checkCudaErrors(hipsolverRfSetNumericProperties(cusolverRfH, nzero, nboost));

    // choose algorithm for refactorization and solve
    checkCudaErrors(hipsolverRfSetAlgs(cusolverRfH, fact_alg, solve_alg));

    // matrix mode: L and U are CSR format, and L has implicit unit diagonal
    checkCudaErrors(hipsolverRfSetMatrixFormat(
        cusolverRfH, HIPSOLVERRF_MATRIX_FORMAT_CSR, HIPSOLVERRF_UNIT_DIAGONAL_ASSUMED_L));

    // fast mode for matrix assembling
    checkCudaErrors(hipsolverRfSetResetValuesFastMode(
        cusolverRfH, HIPSOLVERRF_RESET_VALUES_FAST_MODE_ON));

    printf("step 9: assemble P*A*Q = L*U \n");
    start = second();
    start = second();

    for (int i = 0; i < batchSize; ++i)
    {
        h_A_array[i] = &(h_A_batch[batchSize*i]);
    }
    checkCudaErrors(hipsolverRfBatchSetupHost(
        batchSize,
        rowsA, nnzA, 
        h_csrRowPtrA, h_csrColIndA, h_A_array,
        nnzL, 
        h_csrRowPtrL, h_csrColIndL, h_csrValL, 
        nnzU, 
        h_csrRowPtrU, h_csrColIndU, h_csrValU, 
        h_P, 
        h_Q, 
        cusolverRfH));

    checkCudaErrors(hipDeviceSynchronize());
    stop = second();
    time_rf_assemble = stop - start;

    printf("step 10: analyze to extract parallelism \n");
    checkCudaErrors(hipsolverRfBatchAnalyze(cusolverRfH));

    printf("step 11: import A to cusolverRf \n");
    checkCudaErrors(hipMemcpy(d_csrRowPtrA, h_csrRowPtrA, sizeof(int)*(rowsA+1), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_csrColIndA, h_csrColIndA, sizeof(int)*nnzA     , hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_csrValA   , h_csrValA   , sizeof(double)*nnzA  , hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_A_batch   , h_A_batch  , sizeof(double)*batchSize*nnzA  , hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_P, h_P, sizeof(int)*rowsA, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_Q, h_Q, sizeof(int)*colsA, hipMemcpyHostToDevice));

    start = second();
    start = second();

    checkCudaErrors(hipsolverRfBatchResetValues(
        batchSize,
        rowsA,nnzA,
        d_csrRowPtrA, d_csrColIndA, d_A_array,
        d_P,
        d_Q,
        cusolverRfH));

    checkCudaErrors(hipDeviceSynchronize());
    stop = second();
    time_rf_reset = stop - start;

    printf("step 12: refactorization \n");
    start = second();
    start = second();

    checkCudaErrors(hipsolverRfBatchRefactor(cusolverRfH));

    checkCudaErrors(hipDeviceSynchronize());
    stop = second();
    time_rf_refactor = stop - start;

    printf("step 13: solve A*x = b \n");
    //checkCudaErrors(hipMemcpy(d_x, h_b, sizeof(double)*rowsA, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_X_batch, h_X_batch, sizeof(double)*batchSize*rowsA, hipMemcpyHostToDevice));

    start = second();
    start = second();

    checkCudaErrors(hipsolverRfBatchSolve(cusolverRfH, d_P, d_Q, 1, d_T, rowsA, d_X_array, rowsA));

    checkCudaErrors(hipDeviceSynchronize());
    stop = second();
    time_rf_solve = stop - start;

    printf("step 14: evaluate residual r = b - A*x (result on GPU)\n");
    //checkCudaErrors(hipMemcpy(d_r, h_b, sizeof(double)*rowsA, hipMemcpyHostToDevice));
    //size_t bufferSize = 0;
    
    
    checkCudaErrors(hipsparseSpMV_bufferSize(
        cusparseH, HIPSPARSE_OPERATION_NON_TRANSPOSE, &minus_one, matA, vecx,
        &one, vecAx, HIP_R_64F, HIPSPARSE_CSRMV_ALG1, &bufferSize));
    //void *buffer = NULL;
    cout << "Buffer Size " << bufferSize*1000 << endl;
    checkCudaErrors(hipMalloc(&buffer, bufferSize));
    //checkCudaErrors(hipMemcpy(h_X_batch, d_X_batch, sizeof(double)*batchSize*rowsA, hipMemcpyDeviceToHost));


for (int i=0; i < batchSize; ++i)
    {
        checkCudaErrors(hipMemcpy(d_r, &h_X_batch[i*colsA], sizeof(double)*rowsA, hipMemcpyHostToDevice));
        checkCudaErrors(hipsparseCreateDnVec(&vecx, N, &d_X_batch[i*colsA], HIP_R_64F));
        // todo: hipsparseSpMM
        
        checkCudaErrors(hipsparseSpMV(cusparseH,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &minus_one,
            matA,
            vecx,
            &one,
            vecAx, HIP_R_64F,HIPSPARSE_CSRMV_ALG1, buffer));

        checkCudaErrors(hipMemcpy(h_x, &d_X_batch[i*colsA], sizeof(double)*colsA, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(h_r, d_r, sizeof(double)*rowsA, hipMemcpyDeviceToHost));
        memcpy(&x[i*colsA], h_x, sizeof(double)*colsA);
        //for (int i = 0; i < rowsA; i++)
            //cout << h_x[i] << " " << h_r[i] << endl;
        
        x_inf = vec_norminf(colsA, h_x);
        r_inf = vec_norminf(rowsA, h_r);
        //printf("(GPU) |b - A*x| = %E ", r_inf);
        //printf("(GPU) |A| = %E \n", A_inf);
        //printf("(GPU) |x| = %E \n", x_inf);
        if (i < 10)
        printf("(GPU) |b - A*x|/(|A|*|x|) = %E \n", r_inf/(A_inf * x_inf));
    }
    
    
    printf("===== statistics \n");
    printf(" nnz(A) = %d, nnz(L+U) = %d, zero fill-in ratio = %f\n", 
        nnzA, nnzL + nnzU, ((double)(nnzL+nnzU))/(double)nnzA);
    printf("\n");
    printf("===== timing profile \n");
    printf(" reorder A   : %f sec\n", time_reorder);
    printf(" B = Q*A*Q^T : %f sec\n", time_perm);
    printf("\n");
    printf(" cusolverSp LU analysis: %f sec\n", time_sp_analysis);
    printf(" cusolverSp LU factor  : %f sec\n", time_sp_factor);
    printf(" cusolverSp LU solve   : %f sec\n", time_sp_solve);
    printf(" cusolverSp LU extract : %f sec\n", time_sp_extract);
    printf("\n");
    printf(" cusolverRf assemble : %f sec\n", time_rf_assemble);
    printf(" cusolverRf reset    : %f sec\n", time_rf_reset);
    printf(" cusolverRf refactor : %f sec\n", time_rf_refactor);
    printf(" cusolverRf solve    : %f sec\n", time_rf_solve/batchSize);

    if (cusolverRfH) { checkCudaErrors(hipsolverRfDestroy(cusolverRfH)); }
    if (cusolverSpH) { checkCudaErrors(hipsolverSpDestroy(cusolverSpH)); }
    if (cusparseH  ) { checkCudaErrors(hipsparseDestroy(cusparseH)); }
    if (stream     ) { checkCudaErrors(hipStreamDestroy(stream)); }
    if (descrA     ) { checkCudaErrors(hipsparseDestroyMatDescr(descrA)); }
    if (info       ) { checkCudaErrors(cusolverSpDestroyCsrluInfoHost(info)); }
    if (h_Qreorder  ) { free(h_Qreorder); }
    
    if (h_csrRowPtrB) { free(h_csrRowPtrB); }
    if (h_csrColIndB) { free(h_csrColIndB); }
    if (h_csrValB   ) { free(h_csrValB   ); }
    if (h_mapBfromA ) { free(h_mapBfromA ); }

    if (h_x   ) { free(h_x); }
    if (h_b   ) { free(h_b); }
    if (h_r   ) { free(h_r); }
    if (h_xhat) { free(h_xhat); }
    if (h_bhat) { free(h_bhat); }

    if (buffer_cpu) { free(buffer_cpu); }

    if (h_Plu) { free(h_Plu); }
    if (h_Qlu) { free(h_Qlu); }
    if (h_csrRowPtrL) { free(h_csrRowPtrL); }
    if (h_csrColIndL) { free(h_csrColIndL); }
    if (h_csrValL   ) { free(h_csrValL   ); }
    if (h_csrRowPtrU) { free(h_csrRowPtrU); }
    if (h_csrColIndU) { free(h_csrColIndU); }
    if (h_csrValU   ) { free(h_csrValU   ); }

    if (h_P) { free(h_P); }
    if (h_Q) { free(h_Q); }

    if (d_csrValA   ) { checkCudaErrors(hipFree(d_csrValA)); }
    if (d_csrRowPtrA) { checkCudaErrors(hipFree(d_csrRowPtrA)); }
    if (d_csrColIndA) { checkCudaErrors(hipFree(d_csrColIndA)); }
    if (d_x) { checkCudaErrors(hipFree(d_x)); }
    if (d_b) { checkCudaErrors(hipFree(d_b)); }
    if (d_r) { checkCudaErrors(hipFree(d_r)); }
    if (d_P) { checkCudaErrors(hipFree(d_P)); }
    if (d_Q) { checkCudaErrors(hipFree(d_Q)); }
    if (d_T) { checkCudaErrors(hipFree(d_T)); }

    return 0;
}


